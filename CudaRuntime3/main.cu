#include "hip/hip_runtime.h"
#ifndef __HIPCC__ 
#define __HIPCC__
#endif

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "hip/hip_runtime.h"
#include ""
#include "h_fce.cuh"
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <hipcub/hipcub.hpp>

#include <stdio.h>
#include <time.h>

using namespace std;
using namespace hipcub;






__global__ void searchForBestKernel(volatile float* objectiveValues, size_t* indices)
{
	unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
	__shared__ float oVs[num_of_agents];
	__shared__ unsigned int ind[num_of_agents];

	oVs[id] = objectiveValues[id];
	oVs[id + num_of_agents_half] = objectiveValues[id + num_of_agents_half];
	ind[id] = id;
	ind[id + num_of_agents_half] = id + num_of_agents_half;
	__syncthreads();
	unsigned int step = num_of_agents_half;

#pragma unroll
	for (int i = 0; i < num_of_runs; ++i)
	{
		ind[id] = ((oVs[ind[id + step]] < oVs[ind[id]]) ? ind[id + step] : ind[id]);
		step >>= 1;
		__syncthreads();
	}
	indices[id] = ind[id];
	__syncthreads();
}



 

int main()
{
	//device init
	float* agent_pos = NULL;
	float* agent_val = NULL;
	size_t* indice = NULL;
	int* a = NULL;
	int* b = NULL;
	hiprandGenerator_t pseudo_rand;
	float* best_sol_a = NULL;
	float* y_DE;
	float* y_DE_val;
	float* best_DE;
	size_t* best_de;

	float* agent_best_pso;
	float* agent_best_pso_v;
	size_t* best_pso;

	hipMalloc(&agent_pos, num_of_indices * sizeof(float));
	hipMalloc(&agent_val, num_of_agents * sizeof(float));

	hipMalloc(&y_DE, num_of_indices * sizeof(float));
	hipMalloc(&y_DE_val, num_of_agents * sizeof(float));
	hipMalloc(&best_DE, max_iter * sizeof(float));		//vals
	hipMalloc(&best_de, sizeof(size_t));				//indice

	hipMalloc(&agent_best_pso, num_of_indices * sizeof(float));
	hipMalloc(&agent_best_pso_v, num_of_agents * sizeof(float));
	hipMalloc(&best_pso, max_iter * sizeof(float));		//vals

	hipMalloc(&indice, num_of_agents * sizeof(size_t));
	hipMalloc(&best_sol_a, num_of_best_indices * sizeof(float));
	hipMalloc(&a, num_of_dims * sizeof(int));
	hipMalloc(&b, num_of_dims * sizeof(int));

	//host init
	float* pop_back = NULL;
	float* pop_vals = NULL;
	float* best = NULL;
	size_t* ind;

	pop_back = (float*)malloc(num_of_indices * sizeof(float));
	pop_vals = (float*)malloc(num_of_agents * sizeof(float));
	best = (float*)malloc(max_iter * sizeof(float));
	ind = (size_t*)malloc(num_of_agents * sizeof(size_t));

	// prog
	get_constr << <num_of_dims, 1 >> > (-100, 100, a, b);
	init_pop_pos << <num_of_agents, num_of_dims >> > (agent_pos, a, b, time(NULL));
	cost_func << <num_of_agents, 1 >> > (agent_pos, agent_val);

	hipMemcpy(pop_vals, agent_val, num_of_agents * sizeof(float), ::hipMemcpyDeviceToHost);

	searchForBestKernel << <best_bl_th, best_bl_th >> > (agent_val, indice);
	hipMemcpy(ind, indice, num_of_agents * sizeof(size_t), ::hipMemcpyDeviceToHost);


	hipError_t err; hipError_t err1; hipError_t err2;
	err = hipMemcpy(best_de, &indice[0], sizeof(size_t), ::hipMemcpyDeviceToDevice);

	//DE start
	
	
	//for (int i = 0; i < max_iter; ++i)
	//{
	//	DE << <num_of_agents, num_of_dims >> > (0.4, 0.7, a, b, time(NULL), indice, agent_pos, agent_val, y_DE);
	//	cost_func << <num_of_agents, 1 >> > (y_DE, y_DE_val);
	//	compare_two_pop << <num_of_agents, 1 >> > (agent_pos, agent_val, y_DE, y_DE_val);
	//	searchForBestKernel << <best_bl_th, best_bl_th>> > (agent_val, indice);

	//	hipMemcpy(ind, indice, num_of_agents * sizeof(size_t), ::hipMemcpyDeviceToHost);
	//	err = hipMemcpy(&best[i], &agent_val[ind[0]], sizeof(float), ::hipMemcpyDeviceToHost);
	//	err1 = hipMemcpy(best_de, &indice[0], sizeof(size_t), ::hipMemcpyDeviceToDevice);
	//}

	//DE end


	//PSO start

	agent_best_pso = agent_pos;
	agent_best_pso_v = agent_val;
	for (int i = 0; i < max_iter; ++i)
	{
		pso_f << <num_of_agents, num_of_dims >> > (0.1, 0.25, 2, a, b, time(NULL), indice, agent_pos, agent_best_pso, agent_val);
		cost_func << <num_of_agents, 1 >> > (agent_pos, agent_val);
		compare_two_pop << <num_of_agents, 1 >> > (agent_best_pso, agent_best_pso_v, agent_pos, agent_val);
		searchForBestKernel << <best_bl_th, best_bl_th>> > (agent_best_pso_v, indice);

		hipMemcpy(ind, indice, num_of_agents * sizeof(size_t), ::hipMemcpyDeviceToHost);
		err = hipMemcpy(&best[i], &agent_best_pso_v[ind[0]], sizeof(float), ::hipMemcpyDeviceToHost);

	}

	//PSO end


	//err = hipMemcpy(best, best_DE, max_iter * sizeof(float), ::hipMemcpyDeviceToHost);

	err = hipMemcpy(pop_back, agent_pos, num_of_indices * sizeof(float), ::hipMemcpyDeviceToHost);
	for (int i = 0; i < num_of_agents; ++i)
	{
		cout << i << " ";
		for (int j = 0; j < num_of_dims; ++j)
		{
			cout << pop_back[i * num_of_dims + j] << ", ";
		}
		cout << '\n' << endl;
	}
	cout << '\n' << endl;

	cout << ind[0] << ", " << endl;
	err1 = hipMemcpy(&ind[0], best_de, sizeof(size_t), ::hipMemcpyDeviceToHost);
	cout << ind[0] << ", " << endl;


	for (int i = 0; i < max_iter; ++i)
	{
		cout << best[i] << ", " << endl;
	}
	cout << '\n' << endl;


	free(pop_back);
	free(pop_vals);
	free(best);

	hipFree(agent_pos);
	hipFree(agent_val);
	hipFree(indice);
	hipFree(a);
	hipFree(b);
	hipFree(best_sol_a);
	hipFree(y_DE);
	hipFree(y_DE_val);
	hipFree(best_DE);
	hipFree(best_de);
	

	return 0;
}


//hiprand array
//float* r = NULL;
//hipMalloc(&r, num_of_indices * sizeof(float));
//hiprandCreateGenerator(&pseudo_rand, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
//hiprandSetPseudoRandomGeneratorSeed(pseudo_rand, 1);
//hiprandGenerateUniform(pseudo_rand, r, num_of_indices);
//hiprandGenerateUniform(pseudo_rand, r, num_of_indices);
//hiprandGenerateUniform(pseudo_rand, r, num_of_indices);

