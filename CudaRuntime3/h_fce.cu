#include "hip/hip_runtime.h"
#include"h_fce.cuh"

__global__ void get_constr(const int min, const int max, int* a, int* b)
{
	if (!a || !b)
		return;

	a[blockIdx.x] = min;
	b[blockIdx.x] = max;
	
	return;
}

__global__ void init_pop_pos(float* agent_pos, const int* a, const int* b,unsigned long seed)
{

	int index = threadIdx.x + blockIdx.x * blockDim.x;
	hiprandState state;
	hiprand_init(seed, index, 0, &state);

	agent_pos[index] = (index < num_of_indices) ?
		a[threadIdx.x] + hiprand_uniform(&state) * (b[threadIdx.x] - a[threadIdx.x]) : 0;

}

__global__ void cost_func(const float* agent_pos, float* agent_val)
{
	unsigned int agent = threadIdx.x * num_of_dims + blockIdx.x * num_of_indices;
	unsigned int offset= threadIdx.x + blockIdx.x * num_of_agents;
	//unsigned int agent = blockIdx.x * num_of_dims + blockIdx.y * num_of_indices;
	//unsigned int offset = blockIdx.x + blockIdx.y * num_of_agents;
	agent_val[offset] = 0;
	switch (input_func)
	{
	case 1:
#pragma unroll 
		for (int i = 0; i < num_of_dims; ++i)
		{
			agent_val[offset] += powf(agent_pos[agent + i], 4) - 16 * powf(agent_pos[agent + i], 2)
				+ 5 * agent_pos[agent + i];
		}
		agent_val[offset] /= 2;
		break;

	default:
#pragma unroll 
		for (int i = 0; i < num_of_dims; ++i)
		{
			agent_val[offset] += powf(agent_pos[agent + i], 2);
		}
		break;
	}
}
//__global__ void sphere(const float* agent_pos, float* agent_val)
//{
//	int agent = threadIdx.x * num_of_dims;
//	agent_val[threadIdx.x] = 0;
//	
//#pragma unroll 
//		for (int i = 0; i < num_of_dims; ++i)
//		{
//			agent_val[threadIdx.x] += pow(agent_pos[agent + i], 2);
//		}
//
//}
//__global__ void styblinski�tang(const float* agent_pos, float* agent_val)
//{
//	int agent = threadIdx.x * num_of_dims;
//	agent_val[threadIdx.x] = 0;
//
//#pragma unroll 
//	for (int i = 0; i < num_of_dims; ++i)
//	{
//		agent_val[threadIdx.x] += pow(agent_pos[agent + i], 4) - 16 * pow(agent_pos[agent + i], 2)
//			+ 5 * agent_pos[agent + i];
//	}
//	agent_val[threadIdx.x] /= 2;
//}

__global__ void DE(const float w, const float p, const int* a, const int* b, 
	const unsigned int* Ri, const unsigned int* X, const float* Rj,
	const unsigned int* best_sol, const float* agent_pos, const float* agent_val, float* y)
{
	float u_tmp = 0;
	float u;
	unsigned int index = threadIdx.x + blockIdx.x * num_of_dims;
	unsigned int r_index = 4 * (threadIdx.x + blockIdx.x * num_of_dims);
	unsigned int i_r1, i_r2, i_r3, i_r4;

	i_r1 = (Ri[r_index + 0] % num_of_agents) * num_of_dims + threadIdx.x;
	i_r2 = (Ri[r_index + 1] % num_of_agents) * num_of_dims + threadIdx.x;
	i_r3 = (Ri[r_index + 2] % num_of_agents) * num_of_dims + threadIdx.x;
	i_r4 = (Ri[r_index + 3] % num_of_agents) * num_of_dims + threadIdx.x;

	u_tmp = (index < num_of_indices) ?
		agent_pos[best_sol[0] * blockIdx.x + threadIdx.x] + w * (agent_pos[i_r1] + agent_pos[i_r2] - agent_pos[i_r3] - agent_pos[i_r4])
		//agent_pos[i_r1] + w * (agent_pos[i_r2] - agent_pos[i_r3])
		:
		u_tmp;

	//search dom test
	u = (a[blockIdx.x] <= u_tmp) ? u_tmp : a[blockIdx.x];
	u = (b[blockIdx.x] >= u_tmp) ? u_tmp : b[blockIdx.x];

	//new pos
	y[index] = (Rj[index] <= p || X[index] == blockIdx.x) ? u : agent_pos[index] ;
	//__syncthreads();
}

__global__ void pso_f(const float w, const float c1, const float c2, const int* a, const int* b, const float* r_i,
	const unsigned int* best_sol, float* agent_pos, const float* agent_best_pos, const float* agent_val)
{
	float V = 0;
	float tmp = 0;
	unsigned int index = threadIdx.x + blockIdx.x * num_of_dims;
	unsigned int best_index = threadIdx.x + best_sol[0] * num_of_dims;
	
	unsigned int r2 = index + num_of_indices;
	
	V = w * V + c1 * r_i[index]*(agent_best_pos[index] - agent_pos[index]) + c2 * r_i[r2] 
		* (agent_best_pos[best_index] - agent_pos[index]);

	tmp = V + agent_pos[index];

	agent_pos[index] = (a[threadIdx.x] <= tmp) ? tmp : a[threadIdx.x];
	agent_pos[index] = (b[threadIdx.x] >= tmp) ? tmp : b[threadIdx.x];

	__syncthreads();
}


__global__ void ffa(const float alfa, const float beta, const float gamma, const int* a, const int* b, const float* r,
	 const float* agent_pos, float* agent_new_pos, const float* agent_val)
{
	unsigned int agent_x = blockIdx.x;
	unsigned int agent_y = blockIdx.y;

	unsigned int offset_x = blockIdx.x * blockDim.x;
	unsigned int offset_y = blockIdx.y * blockDim.x;

	unsigned int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int index_y = blockIdx.y * blockDim.x + threadIdx.x;
	unsigned int index_r = agent_x + agent_y * num_of_agents;
	unsigned int index	= index_x + blockDim.x * num_of_agents * agent_y;	// index of pos to save

	float R;
	float tmp;

	if (agent_val[agent_y] < agent_val[agent_x]) {
		R = 0;
#pragma unroll
		for (auto i = 0; i < num_of_dims; i++) {
			R += powf(agent_pos[offset_x + i] - agent_pos[offset_y + i], 2);	//calc distance
		}

		tmp = agent_pos[index_x] + beta * exp(-gamma * R) * (agent_pos[index_x] - agent_pos[index_y])
			+ alfa * r[index_r];		//new pos

		agent_new_pos[index] = (a[threadIdx.x] <= tmp) ? tmp : a[threadIdx.x];
		agent_new_pos[index] = (b[threadIdx.x] >= tmp) ? tmp : b[threadIdx.x];
	}
	else {
			agent_new_pos[index] = agent_pos[index_x];		//save old 
	}
	__syncthreads();
}

__global__ void GWO(const unsigned int* best_ind, const float* r_a, const int* a, const int* b,
	const float A, const float* agent_pos, float* agent_new_pos) 
{
	unsigned int index    = threadIdx.x + blockIdx.x * num_of_dims;
	
	unsigned int A_index  = threadIdx.x + best_ind[0] * num_of_dims;
	unsigned int B_index  = threadIdx.x + best_ind[1] * num_of_dims;
	unsigned int G_index  = threadIdx.x + best_ind[2]* num_of_dims;

	unsigned int rA_index = blockIdx.x;
	unsigned int rB_index = blockIdx.x + num_of_agents;
	unsigned int rG_index = blockIdx.x + 2 * num_of_agents;
	unsigned int r_a_index= 3 * num_of_agents;

	float a_alfa = 2 * A * r_a[rA_index] - A;
	float a_beta = 2 * A * r_a[rB_index] - A;
	float a_gamma= 2 * A * r_a[rG_index] - A;

	float d_alfa = abs(2 * r_a[rA_index + r_a_index] * agent_pos[A_index] - agent_pos[index]);
	float d_beta = abs(2 * r_a[rB_index + r_a_index] * agent_pos[B_index] - agent_pos[index]);
	float d_gamma= abs(2 * r_a[rG_index + r_a_index] * agent_pos[G_index] - agent_pos[index]);

	float X_alfa = agent_pos[A_index] - a_alfa * d_alfa;
	float X_beta = agent_pos[B_index] - a_beta * d_beta;
	float X_gamma= agent_pos[G_index] - a_gamma * d_gamma;

	float tmp = (X_alfa + X_beta + X_gamma) / 3;

	agent_new_pos[index] = (a[threadIdx.x] <= tmp) ? tmp : a[threadIdx.x];
	agent_new_pos[index] = (b[threadIdx.x] >= tmp) ? tmp : b[threadIdx.x];
}

__global__ void iGWO_nh(unsigned int* r_w, const float* r, const int* a, const int* b,
	const float* distance, float* agent_pos, const float* agent_new_pos, float* nh_pos, unsigned int* ind_to_choose)
{
	unsigned int agent = blockIdx.x * num_of_dims;
	unsigned int index = 0;
	unsigned int nh_index = 0;
	unsigned int ind_to_comp = blockIdx.x + blockIdx.x * num_of_agents;	// dist of curr agent n X_gwo
	unsigned int dist_y = blockIdx.x + blockIdx.y * num_of_agents;		// dist of other agents n X_gwo
	unsigned int nh = blockIdx.x + num_of_agents;

	r_w[blockIdx.x] = r_w[blockIdx.x] % num_of_agents;
	r_w[nh] = r_w[nh] % num_of_agents;

	ind_to_choose[dist_y] = (distance[dist_y] <= distance[ind_to_comp]) ? blockIdx.y : blockIdx.x;


//#pragma unroll
//	for (auto i = 0; i < num_of_dims; i++)
//	{
		index = agent + threadIdx.x;
		nh_index = blockIdx.x + r_w[nh] * num_of_agents;
		nh_pos[index] = (ind_to_choose[nh_index] == blockIdx.x) ? //calc X_gwo pos otherwise calc with agent in nh
			agent_pos[index] + r[index] * (agent_new_pos[index] * agent_pos[r_w[blockIdx.x] * num_of_dims + threadIdx.x])
			:
			agent_pos[index] + r[index] * (agent_pos[ind_to_choose[nh_index]] * agent_pos[r_w[blockIdx.x] * num_of_dims + threadIdx.x]);
	//}

	nh_pos[index] = (a[0] <= nh_pos[index]) ? nh_pos[index] : a[0];
	nh_pos[index] = (b[0] >= nh_pos[index]) ? nh_pos[index] : b[0];
}

__global__ void calc_distances(const float* agent_pos, const float* agent_new_pos, float* distance)
{
	unsigned int offset_x = blockIdx.x * num_of_dims;
	unsigned int offset_y = blockIdx.y * num_of_dims;

	unsigned int index = 0;	// index of pos to save

	float R = 0;
	if (blockIdx.x == blockIdx.y)
	{
#pragma unroll
		for (auto i = 0; i < num_of_dims; i++) {
			R += powf(agent_new_pos[offset_x + i] - agent_pos[offset_y + i], 2);	//calc distance
		}
	}
	else
	{
#pragma unroll
		for (auto i = 0; i < num_of_dims; i++) {
			R += powf(agent_pos[offset_x + i] - agent_pos[offset_y + i], 2);	//calc distance
		}
	}
	distance[blockIdx.x + blockIdx.y * num_of_agents] = sqrtf(R);
}

__global__ void iGWO_compare_two_pop(float* pos, float* val, const float* GWO_pos, const float* GWO_val, 
	const float* nh_pos, const float* nh_val)
{
	unsigned int ind = threadIdx.x + blockIdx.x * blockDim.x;

	val[blockIdx.x] = (GWO_val[blockIdx.x] < nh_val[blockIdx.x]) ? GWO_val[blockIdx.x] : nh_val[blockIdx.x];
	pos[ind] = (GWO_val[blockIdx.x] < nh_val[blockIdx.x]) ? GWO_pos[ind] : nh_pos[ind];
}

__global__ void compare_two_pop(float* old_pos, float* old_val, const float* new_pos, const float* new_val)
{
	unsigned int ind = threadIdx.x + blockIdx.x * blockDim.x;

	old_val[blockIdx.x] = (new_val[blockIdx.x] < old_val[blockIdx.x]) ? new_val[blockIdx.x] : old_val[blockIdx.x];
	old_pos[ind] = (new_val[blockIdx.x] < old_val[blockIdx.x]) ? new_pos[ind] : old_pos[ind];
}

__global__ void compare_ff_pos(float* old_pos, float* old_val, const float* new_pos, const float* new_val)
{
	unsigned int ind = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int old_index, new_index;
	unsigned int max_agent = num_of_agents * (num_of_agents-1);

#pragma unroll
	for (auto i = blockIdx.x; i < blockIdx.x + max_agent; i += num_of_agents) {
		
		old_val[blockIdx.x] = (new_val[i] < old_val[blockIdx.x]) ? new_val[i] : old_val[blockIdx.x];
		old_pos[ind] = (new_val[i] <= old_val[blockIdx.x]) ? new_pos[i * num_of_dims + threadIdx.x] : old_pos[ind];
		
//		if (new_val[i] < old_val[threadIdx.x])
//		{
//			old_val[threadIdx.x] = new_val[i];
//			old_index = threadIdx.x * num_of_dims;
//			new_index = i * num_of_dims;
//#pragma unroll
//			for (auto j = 0; j < num_of_dims; ++j)
//			{
//				old_pos[j + old_index] = new_pos[j + new_index];
//			}
//		}
	}
}




















//__global__ void best_sol(const int num_of_agents, const float* agent_val, unsigned int* indice, float* best_val)
//{
//	int j = blockIdx.x * num_of_agents / blockDim.x;
//	best_val[blockIdx.x] = agent_val[j];
//	indice[blockIdx.x] = j;
//
//#pragma unroll
//	for (unsigned int i = j; i < j + num_of_agents / blockDim.x; ++i)
//	{
//		indice[blockIdx.x] = (agent_val[i] < best_val[blockIdx.x]) ? i : indice[blockIdx.x];
//		best_val[blockIdx.x] = (i == indice[blockIdx.x]) ? agent_val[i] : best_val[blockIdx.x];
//	}
//
//}

	//__global__ void ffa(const float alfa, const float beta, const float gamma, const int* a, const int* b, const unsigned long seed,
	//	const unsigned int* best_sol, float* agent_pos, float* agent_new_pos, const float* agent_val)
	//{
	//	unsigned int index_x = threadIdx.x + blockIdx.x * blockDim.x;
	//	unsigned int index_y = threadIdx.x + blockIdx.y * blockDim.y;
	//	unsigned int agent_start = blockIdx.x * blockDim.x*sizeof(double);
	//
	//	__shared__ float R_i[num_of_indices];
	//	__shared__ float  R[num_of_agents];
	//	float  tmp;
	//	float  agent_diff = agent_pos[index] - agent_pos[best_sol[0]];
	//	hiprandState r;
	//	hiprand_init(seed, index, 0, &r);
	//
	//	R_i[index] = pow(agent_diff, 2);
	//	__syncthreads();
	//	
	//	R[blockIdx.x] = thrust::reduce(thrust::device, R_i + agent_start, R_i + agent_start + blockDim.x*sizeof(double));
	//	__syncthreads();
	//
	//	tmp = agent_pos[index] + beta * exp(-gamma * R[blockIdx.x]) * agent_diff + alfa * hiprand_normal(&r);
	//
	//	agent_new_pos[index] = (a[threadIdx.x] <= tmp) ? tmp : a[threadIdx.x];
	//	agent_new_pos[index] = (b[threadIdx.x] >= tmp) ? tmp : b[threadIdx.x];
	//
	//}

//__global__ void ffa(const float alfa, const float beta, const float gamma, const int* a, const int* b, const unsigned long seed,
//	const float* agent_pos, float* agent_new_pos, const float* agent_val)
//{
//	unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
//
//	unsigned int agent_x = blockIdx.x;
//	unsigned int agent_y = threadIdx.x;
//
//	unsigned int offset_x = agent_x * num_of_dims;
//	unsigned int offset_y = agent_y * num_of_dims;
//
//	unsigned int offset_Y = agent_y * (num_of_agents)*num_of_dims;
//
//	unsigned int x_indice;
//
//	float R = 0;
//	float  tmp;
//
//	hiprandState r;
//	hiprand_init(seed, index, 0, &r);
//
//	if (agent_val[agent_y] < agent_val[agent_x]) {
//		R = 0;
//#pragma unroll
//		for (auto i = 0; i < num_of_dims; i++) {
//			R += pow(agent_pos[offset_x + i] - agent_pos[offset_y + i], 2);	//calc distance
//		}
//
//#pragma unroll
//		for (auto i = 0; i < num_of_dims; i++) {
//
//			x_indice = offset_x + i;
//			tmp = agent_pos[x_indice] + beta * exp(-gamma * R) * (agent_pos[x_indice] - agent_pos[offset_y + i])
//				+ alfa * hiprand_normal(&r);		//new pos
//
//			index = x_indice + offset_Y;	//possible pos, 1 column for agent
//			agent_new_pos[index] = (a[threadIdx.x] <= tmp) ? tmp : a[threadIdx.x];
//			agent_new_pos[index] = (b[threadIdx.x] >= tmp) ? tmp : b[threadIdx.x];
//		}
//
//	}
//	else {
//		for (auto i = 0; i < num_of_dims; i++) {
//			x_indice = offset_x + i;
//			index = x_indice + offset_Y;
//			agent_new_pos[index] = agent_pos[x_indice];		//save old 
//		}
//
//	}
//	__syncthreads();
//}
