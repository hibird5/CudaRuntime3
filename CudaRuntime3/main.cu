#ifndef __HIPCC__ 
#define __HIPCC__
#endif

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "hip/hip_runtime.h"
#include ""
#include "h_fce.cuh"
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <hipcub/hipcub.hpp>

#include <stdio.h>
#include <time.h>

using namespace std;
using namespace hipcub;

#define num_of_agents 150
#define num_of_dims 3
#define num_of_indices 450
#define input_func 2
#define num_of_best_indices 50


int main()
{
	//device init
	float* agent_pos = NULL;
	float* agent_val = NULL;
	size_t* indice = NULL;
	int* a = NULL;
	int* b = NULL;
	float* r = NULL;
	hiprandGenerator_t pseudo_rand;
	float* best_sol_a = NULL;

	hipMalloc(&agent_pos, num_of_indices * sizeof(float));
	hipMalloc(&agent_val, num_of_agents * sizeof(float));
	hipMalloc(&indice, num_of_best_indices* sizeof(size_t));
	hipMalloc(&best_sol_a, num_of_best_indices * sizeof(float));
	hipMalloc(&a, num_of_dims * sizeof(int));
	hipMalloc(&b, num_of_dims * sizeof(int));
	hipMalloc(&r, num_of_indices * sizeof(float));

	hiprandCreateGenerator(&pseudo_rand, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
	hiprandSetPseudoRandomGeneratorSeed(pseudo_rand, 1);
	hiprandGenerateUniform(pseudo_rand, r, num_of_indices);
	hiprandGenerateUniform(pseudo_rand, r, num_of_indices);
	hiprandGenerateUniform(pseudo_rand, r, num_of_indices);
	//host init
	float* pop_back = NULL;
	float* pop_vals = NULL;
	float* best = NULL;
	size_t* ind;

	pop_back = (float*)malloc(num_of_indices * sizeof(float));
	pop_vals = (float*)malloc(num_of_agents * sizeof(float));
	best = (float*)malloc(num_of_best_indices * sizeof(float));
	ind = (size_t*)malloc(num_of_best_indices* sizeof(size_t));

	// prog
	get_constr <<<num_of_dims, 1 >>> (-100, 100, a, b);
	init_pop_pos <<<num_of_agents, num_of_dims >>> (agent_pos, num_of_indices, a, b, r);
	cost_func <<<num_of_agents, 1 >>> (num_of_dims, agent_pos, input_func, agent_val);
	best_sol<<<num_of_best_indices,1>>>(num_of_agents, agent_val, indice, best_sol_a);



	hipMemcpy(ind, indice, sizeof(size_t), ::hipMemcpyDeviceToHost);
	hipMemcpy(best, best_sol_a, sizeof(float), ::hipMemcpyDeviceToHost);
	hipMemcpy(pop_back, agent_pos, num_of_indices * sizeof(float), ::hipMemcpyDeviceToHost);
	hipMemcpy(pop_vals, agent_val, num_of_agents * sizeof(float), ::hipMemcpyDeviceToHost);
	
	for (unsigned int i = 0; i < num_of_best_indices; ++i)
	{
		cout << best[i] << ", ";
		//cout << rand() << ", ";
	}
	cout << '\n' << endl;


	for (int i = 0; i < num_of_agents; ++i)
	{
		for (int j = 0; j < num_of_dims; ++j)
		{
			cout << pop_back[i * num_of_dims + j] << ", ";
			//		cout << rand() << ", ";
		}
		cout << '\n' << endl;
	}

	for (int i = 0; i < num_of_agents; ++i)
	{
		cout << pop_vals[i] << ", ";
	}
	cout << '\n' << endl;


	return 0;
}

