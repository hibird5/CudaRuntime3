
#include"alg.cuh"


 __host__ void Diff_ev(const float w, const float p, const float* init_pop,const float* init_vals,const int* a,const  int* b,
	float* best_pos, float* best_vals, float time_per_iter) {

	float* agent_pos = NULL;			
	float* agent_val = NULL;
	float* y_DE_pos = NULL;
	float* y_DE_val = NULL;

	unsigned int* best_index = NULL;
	unsigned int* r = NULL;
	unsigned int* X = NULL;
	unsigned int* HOST_best_index = (unsigned int*)malloc(num_of_agents_half * sizeof(unsigned int)); 
	float* Rj;


	hipError_t err = (hipError_t)0;
	hiprandGenerator_t r_int;
	unsigned int num_of_Ri = 4 * num_of_indices;
	
	error_h(hipGetLastError());

	hiprandCreateGenerator(&r_int, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
	hiprandStatus_t aaa =  hiprandSetPseudoRandomGeneratorSeed(r_int, 5);

	hipMalloc(&agent_pos, num_of_indices * sizeof(float));
	hipMalloc(&agent_val, num_of_agents * sizeof(float));
	hipMalloc(&y_DE_pos, num_of_indices * sizeof(float));
	hipMalloc(&y_DE_val, num_of_agents * sizeof(float));
	hipMalloc(&r, num_of_Ri * sizeof(unsigned int));
	hipMalloc(&X, num_of_indices * sizeof(unsigned int));
	hipMalloc(&Rj, num_of_indices * sizeof(float));
	hipMalloc(&best_index, num_of_agents_half* sizeof(unsigned int));

	err = hipGetLastError();

	hipMemcpy(agent_pos, init_pop, num_of_indices * sizeof(float), ::hipMemcpyDeviceToDevice);
	hipMemcpy(agent_val, init_vals, num_of_agents * sizeof(float), ::hipMemcpyDeviceToDevice);
	
	auto s = std::chrono::high_resolution_clock::now();
	long long iter_time = 0;

	for (int i = 0; i < max_iter; ++i)
	{
		s = std::chrono::high_resolution_clock::now();
		
		//init rands
		hiprandGenerate(r_int, r, num_of_Ri);
		hiprandGenerate(r_int, X, num_of_indices);
		hiprandGenerateUniform(r_int, Rj, num_of_indices);
		
		//calc new pos
		DE <<<num_of_agents, num_of_dims>>> (w, p, a, b, r, X, Rj, best_index, agent_pos, agent_val, y_DE_pos);
		cost_func <<<num_of_agents, dims_to_log_half >>> (y_DE_pos, y_DE_val);
		compare_two_pop <<<num_of_agents, num_of_dims >>> (agent_pos, agent_val, y_DE_pos, y_DE_val);

		//find best pos
		hipDeviceSynchronize();
		searchForBestKernel <<<1, num_of_agents_half>>> (agent_val, best_index);
	
		iter_time += std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::high_resolution_clock::now() - s).count();
		
		hipMemcpy(HOST_best_index, best_index, num_of_agents_half * sizeof(unsigned int), ::hipMemcpyDeviceToHost);
		hipMemcpy(&best_vals[i], &agent_val[HOST_best_index[0]], sizeof(float), ::hipMemcpyDeviceToDevice);
		
		err = hipGetLastError();
		
		if (err != 0)
			break;
	}

	hipMemcpy(best_pos, &agent_pos[HOST_best_index[0]], num_of_dims * sizeof(float), ::hipMemcpyDeviceToDevice);
	time_per_iter = iter_time / max_iter;

	hipFree(agent_pos);
	hipFree(agent_val);
	hipFree(y_DE_pos);
	hipFree(y_DE_val);
	hipFree(r);
	hipFree(X);
	hipFree(Rj);

	error_h(err);
}

__host__ void PSO(const float w, const float c1, const float c2, const float* init_pop, const float* init_vals, const int* a, const int* b,
	float* best_pos, float* best_vals, float time_per_iter) {
	
	float* agent_pos = NULL;
	float* agent_best_pos = NULL;
	float* agent_val = NULL;
	float* agent_best_val = NULL;

	unsigned int* best_index = NULL;
	float* r;
	unsigned int num_of_ri = 2 * num_of_indices;
	unsigned int* HOST_best_index = (unsigned int*)malloc(num_of_agents_half * sizeof(unsigned int));


	hipMalloc(&agent_pos, num_of_indices * sizeof(float));
	hipMalloc(&agent_best_pos, num_of_indices * sizeof(float));
	hipMalloc(&agent_val, num_of_agents * sizeof(float));
	hipMalloc(&agent_best_val, num_of_agents * sizeof(float));
	hipMalloc(&r, num_of_ri * sizeof(float));
	hipMalloc(&best_index, num_of_agents_half * sizeof(unsigned int));

	hipError_t err = (hipError_t)0;
	hiprandGenerator_t r_int;
	hiprandCreateGenerator(&r_int, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
	hiprandSetPseudoRandomGeneratorSeed(r_int, time(NULL));

	hipMemcpy(agent_pos, init_pop, num_of_indices * sizeof(float), ::hipMemcpyDeviceToDevice);
	hipMemcpy(agent_best_pos, init_pop, num_of_indices * sizeof(float), ::hipMemcpyDeviceToDevice);
	hipMemcpy(agent_val, init_vals, num_of_agents * sizeof(float), ::hipMemcpyDeviceToDevice);
	hipMemcpy(agent_best_val, init_vals, num_of_agents * sizeof(float), ::hipMemcpyDeviceToDevice);

	auto s = std::chrono::high_resolution_clock::now();
	long long iter_time = 0;

	for (int i = 0; i < max_iter; ++i)
	{
		s = std::chrono::high_resolution_clock::now();

		//init rand
		hiprandGenerateUniform(r_int, r, num_of_ri);
		
		//calc new pos
		pso_f << <  num_of_agents, num_of_dims >> > (w, c1, c2, a, b, r, best_index, agent_pos, agent_best_pos, agent_val); //V --
		cost_func << <num_of_agents, dims_to_log_half >> > (agent_pos, agent_val);
		compare_two_pop << <num_of_agents, num_of_dims >> > (agent_best_pos, agent_best_val, agent_pos, agent_val);

		//find best pos
		hipDeviceSynchronize();
		searchForBestKernel << <1, num_of_agents_half >> > (agent_best_val, best_index);
		
		iter_time += std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::high_resolution_clock::now() - s).count();

		hipMemcpy(HOST_best_index, best_index, num_of_agents_half * sizeof(unsigned int), ::hipMemcpyDeviceToHost);
		hipMemcpy(&best_vals[i], &agent_best_val[HOST_best_index[0]], sizeof(float), ::hipMemcpyDeviceToHost);

		err = hipGetLastError();

		if (err != 0)
			break;
	}

	hipMemcpy(best_pos, &agent_pos[HOST_best_index[0]], num_of_dims * sizeof(float), ::hipMemcpyDeviceToDevice);
	time_per_iter = iter_time / max_iter;

	hipFree(agent_pos);
	hipFree(agent_best_pos);
	hipFree(agent_val);
	hipFree(agent_best_val);
	hipFree(r);
	hipFree(best_index);

	error_h(err);
}

__host__ void FF(const float* init_pop, const float* init_vals, const int* a, const int* b, 
	float* best_pos, float* best_vals, float time_per_iter) {

	float* agent_pos = NULL;
	float* agent_val = NULL;

	hipMalloc(&agent_pos, num_of_indices * sizeof(float));
	hipMalloc(&agent_val, num_of_agents * sizeof(float));
}

__host__ void ABC(const float* init_pop,const float* init_vals, const int* a, const int* b, 
	float* best_pos, float* best_vals, float time_per_iter) {

	float* agent_pos = NULL;
	float* agent_val = NULL;

	hipMalloc(&agent_pos, num_of_indices * sizeof(float));
	hipMalloc(&agent_val, num_of_agents * sizeof(float));

}

__host__ void GWO(const float* init_pop, const float* init_vals, const int* a, const int* b, 
	float* best_pos, float* best_vals, float time_per_iter) {

	float* agent_pos = NULL;
	float* agent_val = NULL;

	hipMalloc(&agent_pos, num_of_indices * sizeof(float));
	hipMalloc(&agent_val, num_of_agents * sizeof(float));

}

__host__ void iGWO(const float* init_pop, const float* init_vals, const int* a, const int* b, 
	float* best_pos, float* best_vals, float time_per_iter) {

	float* agent_pos = NULL;
	float* agent_val = NULL;

	hipMalloc(&agent_pos, num_of_indices * sizeof(float));
	hipMalloc(&agent_val, num_of_agents * sizeof(float));

}