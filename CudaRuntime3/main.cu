#include "hip/hip_runtime.h"
#ifndef __HIPCC__ 
#define __HIPCC__
#endif

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "hip/hip_runtime.h"
#include ""
#include "h_fce.cuh"
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <hipcub/hipcub.hpp>

#include <stdio.h>
#include <time.h>

using namespace std;
using namespace hipcub;


#define num_of_agents 150
#define num_of_dims 3
#define num_of_indices 450
#define input_func 1
#define num_of_best_indices 50


int main()
{
	//device init
	float* agent_pos = NULL;
	float* agent_val = NULL;
	size_t* indice = NULL;
	int* a = NULL;
	int* b = NULL;
	hiprandGenerator_t pseudo_rand;
	float* best_sol_a = NULL;

	hipMalloc(&agent_pos, num_of_indices * sizeof(float));
	hipMalloc(&agent_val, num_of_agents * sizeof(float));
	hipMalloc(&indice, num_of_best_indices* sizeof(size_t));
	hipMalloc(&best_sol_a, num_of_best_indices * sizeof(float));
	hipMalloc(&a, num_of_dims * sizeof(int));
	hipMalloc(&b, num_of_dims * sizeof(int));

	//host init
	float* pop_back = NULL;
	float* pop_vals = NULL;
	float* best = NULL;
	size_t* ind;

	pop_back = (float*)malloc(num_of_indices * sizeof(float));
	pop_vals = (float*)malloc(num_of_agents * sizeof(float));
	best = (float*)malloc(num_of_best_indices * sizeof(float));
	ind = (size_t*)malloc(num_of_best_indices* sizeof(size_t));

	// prog
	get_constr <<<num_of_dims, 1 >>> (-100, 100, a, b);
	init_pop_pos <<<num_of_agents, num_of_dims >>> (agent_pos, num_of_indices, a, b, time(NULL));
	cost_func <<<num_of_agents, 1 >>> (num_of_dims, agent_pos, input_func, agent_val);
	best_sol<<<num_of_best_indices,1>>>(num_of_agents, agent_val, indice, best_sol_a);

	
	hipMemcpy(ind, indice, sizeof(size_t), ::hipMemcpyDeviceToHost);
	hipMemcpy(best, best_sol_a, sizeof(float), ::hipMemcpyDeviceToHost);
	hipMemcpy(pop_back, agent_pos, num_of_indices * sizeof(float), ::hipMemcpyDeviceToHost);
	hipMemcpy(pop_vals, agent_val, num_of_agents * sizeof(float), ::hipMemcpyDeviceToHost);
	
	//hiprandState r1, r2, r3;
	//hiprand_init(1, 1, 0, &r1);
	////hiprand_init(seed, blockIdx.x, 0, &state);
	////hiprand_init(seed, blockIdx.x, 0, &state);
	//hiprand_uniform(&r1);
	//cout << (int)r1 << endl;

	//for (unsigned int i = 0; i < num_of_best_indices; ++i)
	//{
		//cout << rand() << ", ";
	//}
	//cout << '\n' << endl;


	for (int i = 0; i < num_of_agents; ++i)
	{
		for (int j = 0; j < num_of_dims; ++j)
		{
			cout << pop_back[i * num_of_dims + j] << ", ";
		}
		cout << '\n' << endl;
	}
	cout << '\n' << endl;

	for (int i = 0; i < num_of_agents; ++i)
	{
		cout << pop_vals[i] << ", ";
	}
	cout << '\n' << endl;

	cout << best[0] << ", " << ind[0]<< endl;

	return 0;
}


//hiprand array
//float* r = NULL;
//hipMalloc(&r, num_of_indices * sizeof(float));
//hiprandCreateGenerator(&pseudo_rand, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
//hiprandSetPseudoRandomGeneratorSeed(pseudo_rand, 1);
//hiprandGenerateUniform(pseudo_rand, r, num_of_indices);
//hiprandGenerateUniform(pseudo_rand, r, num_of_indices);
//hiprandGenerateUniform(pseudo_rand, r, num_of_indices);

__global__ void DE(const float w, const float p, const int* a, const int* b,const unsigned long seed, const unsigned int best_sol,
	float* agent_pos, float* agent_val)
{
	//init	
	__shared__ float u[num_of_indices];
	float u_tmp = 0;
	unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int i_r1, i_r2, i_r3, i_r4;
	hiprandState r1; // , r2, r3;
	hiprand_init(seed, blockIdx.x, 0, &r1);
	//hiprand_init(seed, blockIdx.x, 1, &r2);
	//hiprand_init(seed, blockIdx.x, 2, &r3);
	i_r1 = threadIdx.x + (hiprand(&r1) % blockDim.x) * blockDim.x;
	i_r2 = threadIdx.x + (hiprand(&r1) % blockDim.x) * blockDim.x;
	i_r3 = threadIdx.x + (hiprand(&r1) % blockDim.x) * blockDim.x;
	i_r4 = threadIdx.x + (hiprand(&r1) % blockDim.x) * blockDim.x;

	u_tmp = (index < num_of_indices) ?
		agent_pos[best_sol + threadIdx.x] + w * (agent_pos[i_r1] + agent_pos[i_r2] - agent_pos[i_r3] - agent_pos[i_r4]) 
		: 
		u_tmp;

	u[index] = (a[threadIdx.x] <= u_tmp) ? u_tmp : a[threadIdx.x];
	u[index] = (b[threadIdx.x] >= u_tmp) ? u_tmp : b[threadIdx.x];

	/*
	rj
	x
	if 
	...


	best pos ?
	*/



}