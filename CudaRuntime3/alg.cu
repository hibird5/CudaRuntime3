#include"alg.cuh"

__host__  void Diff_ev(const float w, const float p, const float* init_pop,const float* init_vals,const int* a,const  int* b,
	float* best_pos, float* best_vals, float time_per_iter) {

	float* agent_pos = NULL;
	float* agent_val = NULL;
	float* y_DE_pos = NULL;
	float* y_DE_val = NULL;

	unsigned int* best_index = NULL;
	unsigned int* r;
	unsigned int* X;
	unsigned int* best_ind = (unsigned int*)malloc(num_of_agents_half * sizeof(unsigned int)); ;
	float* Rj;


	hipError_t err = (hipError_t)0;
	hiprandGenerator_t r_int;
	unsigned int num_of_Ri = 4 * num_of_indices;

	hipMalloc(&agent_pos, num_of_indices * sizeof(float));
	hipMalloc(&agent_val, num_of_agents * sizeof(float));
	hipMalloc(&y_DE_pos, num_of_indices * sizeof(float));
	hipMalloc(&y_DE_val, num_of_agents * sizeof(float));
	hipMalloc(&r, num_of_Ri * sizeof(unsigned int));
	hipMalloc(&X, num_of_indices * sizeof(unsigned int));
	hipMalloc(&Rj, num_of_indices * sizeof(float));
	hipMalloc(&best_index, num_of_agents_half* sizeof(unsigned int));

	hiprandCreateGenerator(&r_int, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
	err = hipGetLastError();
	hiprandStatus_t aaa =  hiprandSetPseudoRandomGeneratorSeed(r_int, 5);
	//err = hipGetLastError();
	
	//auto s = 0; // std::chrono::high_resolution_clock::now();
	//long long iter_time ;

	for (int i = 0; i < max_iter; ++i)
	{
		//s = std::chrono::high_resolution_clock::now();
		
		//init rands
		hiprandGenerate(r_int, r, num_of_Ri);
		hiprandGenerate(r_int, X, num_of_indices);
		hiprandGenerateUniform(r_int, Rj, num_of_indices);
		
		//calc new pos
		DE <<<num_of_agents, num_of_dims>>> (w, p, a, b, r, X, Rj, best_index, agent_pos, agent_val, y_DE_pos);
		cost_func <<<num_of_agents, dims_to_log_half >>> (y_DE_pos, y_DE_val);
		compare_two_pop <<<num_of_agents, num_of_dims >>> (agent_pos, agent_val, y_DE_pos, y_DE_val);

		//find best pos
		hipDeviceSynchronize();
		searchForBestKernel <<<1, num_of_agents_half>>> (agent_val, best_index);
	
		
		err = hipGetLastError();
		hipMemcpy(&best_vals[i], &agent_val[best_index[0]], sizeof(float), ::hipMemcpyDeviceToDevice);
		//iter_time += std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::high_resolution_clock::now() - s).count();
		if (err != 0)
			break;
	}

	hipMemcpy(best_pos, &agent_pos[best_index[0]], num_of_dims * sizeof(float), ::hipMemcpyDeviceToDevice);
	//time_per_iter = iter_time / max_iter;

	hipFree(agent_pos);
	hipFree(agent_val);
	hipFree(y_DE_pos);
	hipFree(y_DE_val);
	hipFree(r);
	hipFree(X);
	hipFree(Rj);

	error_h(err);
}

__host__ void PSO(const float* init_pop, const float* init_vals, const int* a, const int* b, 
	float* best_pos, float* best_vals, float time_per_iter) {
	
	float* agent_pos = NULL;
	float* agent_val = NULL;

	hipMalloc(&agent_pos, num_of_indices * sizeof(float));
	hipMalloc(&agent_val, num_of_agents * sizeof(float));
}

__host__ void FF(const float* init_pop, const float* init_vals, const int* a, const int* b, 
	float* best_pos, float* best_vals, float time_per_iter) {

	float* agent_pos = NULL;
	float* agent_val = NULL;

	hipMalloc(&agent_pos, num_of_indices * sizeof(float));
	hipMalloc(&agent_val, num_of_agents * sizeof(float));
}

__host__ void ABC(const float* init_pop,const float* init_vals, const int* a, const int* b, 
	float* best_pos, float* best_vals, float time_per_iter) {

	float* agent_pos = NULL;
	float* agent_val = NULL;

	hipMalloc(&agent_pos, num_of_indices * sizeof(float));
	hipMalloc(&agent_val, num_of_agents * sizeof(float));

}

__host__ void GWO(const float* init_pop, const float* init_vals, const int* a, const int* b, 
	float* best_pos, float* best_vals, float time_per_iter) {

	float* agent_pos = NULL;
	float* agent_val = NULL;

	hipMalloc(&agent_pos, num_of_indices * sizeof(float));
	hipMalloc(&agent_val, num_of_agents * sizeof(float));

}

__host__ void iGWO(const float* init_pop, const float* init_vals, const int* a, const int* b, 
	float* best_pos, float* best_vals, float time_per_iter) {

	float* agent_pos = NULL;
	float* agent_val = NULL;

	hipMalloc(&agent_pos, num_of_indices * sizeof(float));
	hipMalloc(&agent_val, num_of_agents * sizeof(float));

}