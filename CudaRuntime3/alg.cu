
#include"alg.cuh"


 __host__ void Diff_ev(const float w, const float p, const float* init_pop,const float* init_vals,const int* a,const  int* b,
	float* best_pos, float* best_vals, float time_per_iter) {
	
	//Initialization
	float* agent_pos = NULL;			
	float* agent_val = NULL;
	float* y_DE_pos = NULL;
	float* y_DE_val = NULL;

	unsigned int* best_index = NULL;
	unsigned int* r = NULL;
	unsigned int* X = NULL;
	unsigned int* HOST_best_index = (unsigned int*)malloc(NUM_OF_AGENTS_HALF * sizeof(unsigned int)); 
	float* Rj;

	hipError_t err = (hipError_t)0;
	hiprandGenerator_t r_gen;
	unsigned int num_of_Ri = 4 * NUM_OF_INDICES;

	hiprandCreateGenerator(&r_gen, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
	hiprandSetPseudoRandomGeneratorSeed(r_gen, 5);

	hipMalloc(&agent_pos, NUM_OF_INDICES * sizeof(float));
	hipMalloc(&agent_val, NUM_OF_AGENTS * sizeof(float));
	hipMalloc(&y_DE_pos, NUM_OF_INDICES * sizeof(float));
	hipMalloc(&y_DE_val, NUM_OF_AGENTS * sizeof(float));
	hipMalloc(&r, num_of_Ri * sizeof(unsigned int));
	hipMalloc(&X, NUM_OF_INDICES * sizeof(unsigned int));
	hipMalloc(&Rj, NUM_OF_INDICES * sizeof(float));
	hipMalloc(&best_index, NUM_OF_AGENTS_HALF* sizeof(unsigned int));

	err = hipGetLastError();

	hipMemcpy(agent_pos, init_pop, NUM_OF_INDICES * sizeof(float), ::hipMemcpyDeviceToDevice);
	hipMemcpy(agent_val, init_vals, NUM_OF_AGENTS * sizeof(float), ::hipMemcpyDeviceToDevice);
	
	error_h(hipGetLastError());

	auto s = std::chrono::high_resolution_clock::now();
	long long iter_time = 0;

	//Loop
	for (int i = 0; i < MAX_ITER; ++i)
	{
		s = std::chrono::high_resolution_clock::now();
		
		//init rands
		hiprandGenerate(r_gen, r, num_of_Ri);
		hiprandGenerate(r_gen, X, NUM_OF_INDICES);
		hiprandGenerateUniform(r_gen, Rj, NUM_OF_INDICES);
		
		//calc new pos
		DE <<<NUM_OF_AGENTS, NUM_OF_DIMS>>> (w, p, a, b, r, X, Rj, best_index, agent_pos, agent_val, y_DE_pos);
		cost_func <<<NUM_OF_AGENTS, DIMS_TO_LOG_HALF >>> (y_DE_pos, y_DE_val);
		compare_two_pop <<<NUM_OF_AGENTS, NUM_OF_DIMS >>> (agent_pos, agent_val, y_DE_pos, y_DE_val);

		//find best pos
		hipDeviceSynchronize();
		searchForBestKernel <<<1, NUM_OF_AGENTS_HALF>>> (agent_val, best_index);
	
		iter_time += std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::high_resolution_clock::now() - s).count();
		
		hipMemcpy(HOST_best_index, best_index, NUM_OF_AGENTS_HALF * sizeof(unsigned int), ::hipMemcpyDeviceToHost);
		hipMemcpy(&best_vals[i], &agent_val[HOST_best_index[0]], sizeof(float), ::hipMemcpyDeviceToDevice);
		
		err = hipGetLastError();
		
		if (err != 0)
			break;
	}

	hipMemcpy(best_pos, &agent_pos[HOST_best_index[0]], NUM_OF_DIMS * sizeof(float), ::hipMemcpyDeviceToDevice);
	time_per_iter = iter_time / MAX_ITER;

	hipFree(agent_pos);
	hipFree(agent_val);
	hipFree(y_DE_pos);
	hipFree(y_DE_val);
	hipFree(r);
	hipFree(X);
	hipFree(Rj);

	error_h(err);
}

__host__ void PSO(const float w, const float c1, const float c2, const float* init_pop, const float* init_vals, const int* a, const int* b,
	float* best_pos, float* best_vals, float time_per_iter) {
	
	//Initialization
	float* agent_pos = NULL;
	float* agent_best_pos = NULL;
	float* agent_val = NULL;
	float* agent_best_val = NULL;

	unsigned int* best_index = NULL;
	float* r;
	unsigned int num_of_ri = 2 * NUM_OF_INDICES;
	unsigned int* HOST_best_index = (unsigned int*)malloc(NUM_OF_AGENTS_HALF * sizeof(unsigned int));

	hipError_t err = (hipError_t)0;
	hiprandGenerator_t r_gen;
	hiprandCreateGenerator(&r_gen, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
	hiprandSetPseudoRandomGeneratorSeed(r_gen, time(NULL));

	hipMalloc(&agent_pos, NUM_OF_INDICES * sizeof(float));
	hipMalloc(&agent_best_pos, NUM_OF_INDICES * sizeof(float));
	hipMalloc(&agent_val, NUM_OF_AGENTS * sizeof(float));
	hipMalloc(&agent_best_val, NUM_OF_AGENTS * sizeof(float));
	hipMalloc(&r, num_of_ri * sizeof(float));
	hipMalloc(&best_index, NUM_OF_AGENTS_HALF * sizeof(unsigned int));


	hipMemcpy(agent_pos, init_pop, NUM_OF_INDICES * sizeof(float), ::hipMemcpyDeviceToDevice);
	hipMemcpy(agent_best_pos, init_pop, NUM_OF_INDICES * sizeof(float), ::hipMemcpyDeviceToDevice);
	hipMemcpy(agent_val, init_vals, NUM_OF_AGENTS * sizeof(float), ::hipMemcpyDeviceToDevice);
	hipMemcpy(agent_best_val, init_vals, NUM_OF_AGENTS * sizeof(float), ::hipMemcpyDeviceToDevice);


	auto s = std::chrono::high_resolution_clock::now();
	long long iter_time = 0;

	//Loop
	for (int i = 0; i < MAX_ITER; ++i)
	{
		s = std::chrono::high_resolution_clock::now();

		//init rand
		hiprandGenerateUniform(r_gen, r, num_of_ri);
		
		//calc new pos
		pso_f <<<NUM_OF_AGENTS, NUM_OF_DIMS >>> (w, c1, c2, a, b, r, best_index, agent_pos, agent_best_pos, agent_val); //V --
		cost_func <<<NUM_OF_AGENTS, DIMS_TO_LOG_HALF >>> (agent_pos, agent_val);
		compare_two_pop <<<NUM_OF_AGENTS, NUM_OF_DIMS >>> (agent_best_pos, agent_best_val, agent_pos, agent_val);

		//find best pos
		hipDeviceSynchronize();
		searchForBestKernel << <1, NUM_OF_AGENTS_HALF >> > (agent_best_val, best_index);
		
		iter_time += std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::high_resolution_clock::now() - s).count();

		hipMemcpy(HOST_best_index, best_index, NUM_OF_AGENTS_HALF * sizeof(unsigned int), ::hipMemcpyDeviceToHost);
		hipMemcpy(&best_vals[i], &agent_best_val[HOST_best_index[0]], sizeof(float), ::hipMemcpyDeviceToHost);

		err = hipGetLastError();

		if (err != 0)
			break;
	}

	hipMemcpy(best_pos, &agent_pos[HOST_best_index[0]], NUM_OF_DIMS * sizeof(float), ::hipMemcpyDeviceToDevice);
	time_per_iter = iter_time / MAX_ITER;

	hipFree(agent_pos);
	hipFree(agent_best_pos);
	hipFree(agent_val);
	hipFree(agent_best_val);
	hipFree(r);
	hipFree(best_index);

	error_h(err);
}

__host__ void FF(const float alfa, const float beta, const float gamma, const float* init_pop, const float* init_vals, const int* a, const int* b,
	float* best_pos, float* best_vals, float time_per_iter) {

	//Initialization
	float* agent_pos = NULL;
	float* agent_val = NULL;
	float* new_pos = NULL;
	float* new_vals = NULL;
	float* cost_func_tmp = NULL;

	unsigned int* best_index = NULL;
	unsigned int* HOST_best_index = (unsigned int*)malloc(NUM_OF_AGENTS_HALF * sizeof(unsigned int));
	float* r;
	unsigned int num_of_uR = POW_OF_AGENTS * NUM_OF_DIMS;

	dim3 agents(NUM_OF_AGENTS, NUM_OF_AGENTS, 1);
	hipError_t err = (hipError_t)0;
	hiprandGenerator_t r_gen;
	hiprandCreateGenerator(&r_gen, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
	hiprandSetPseudoRandomGeneratorSeed(r_gen, time(NULL));
	
	hipMalloc(&agent_pos, NUM_OF_INDICES * sizeof(float));
	hipMalloc(&agent_val, NUM_OF_AGENTS * sizeof(float));
	hipMalloc(&best_index, NUM_OF_AGENTS_HALF * sizeof(unsigned int));
	hipMalloc(&r, num_of_uR * sizeof(float));
	hipMalloc(&new_pos, NUM_OF_AGENTS * NUM_OF_INDICES * sizeof(float));
	hipMalloc(&new_vals, NUM_OF_AGENTS * NUM_OF_AGENTS * sizeof(float));
	hipMalloc(&cost_func_tmp, POW_OF_AGENTS * DIMS_TO_LOG_HALF * sizeof(float));

	hipMemcpy(agent_pos, init_pop, NUM_OF_INDICES * sizeof(float), ::hipMemcpyDeviceToDevice);
	hipMemcpy(agent_val, init_vals, NUM_OF_AGENTS * sizeof(float), ::hipMemcpyDeviceToDevice);

	auto s = std::chrono::high_resolution_clock::now();
	long long iter_time = 0;

	for (int i = 0; i < MAX_ITER; ++i)
	{
		s = std::chrono::high_resolution_clock::now();
		//init rand
		hiprandGenerateNormal(r_gen, r, num_of_uR, 0.0, 0.5);

		//calc new pos
		ffa << <agents, NUM_OF_DIMS >> > (alfa, beta, gamma, a, b, r, agent_pos, new_pos, agent_val);
		cost_func << <agents, DIMS_TO_LOG_HALF >> > (new_pos, new_vals, cost_func_tmp);
		compare_ff_pop << <NUM_OF_AGENTS, NUM_OF_DIMS >> > (agent_pos, agent_val, new_pos, new_vals);

		//find best sol
		hipDeviceSynchronize();
		searchForBestKernel << <1, NUM_OF_AGENTS_HALF >> > (agent_val, best_index);
		
		iter_time += std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::high_resolution_clock::now() - s).count();

		hipMemcpy(HOST_best_index, best_index, NUM_OF_AGENTS_HALF * sizeof(unsigned int), ::hipMemcpyDeviceToHost);
		hipMemcpy(&best_vals[i], &agent_val[HOST_best_index[0]], sizeof(float), ::hipMemcpyDeviceToHost);

		err = hipGetLastError();

		if (err != 0)
			break;

	}

	hipMemcpy(best_pos, &agent_pos[HOST_best_index[0]], NUM_OF_DIMS * sizeof(float), ::hipMemcpyDeviceToDevice);
	time_per_iter = iter_time / MAX_ITER;

	hipFree(agent_pos);
	hipFree(agent_val);
	hipFree(best_index);
	hipFree(r);
	hipFree(new_pos);
	hipFree(new_vals);
	hipFree(cost_func_tmp);

	error_h(err);
}

__host__ void ABC(const float* init_pop,const float* init_vals, const int* a, const int* b, 
	float* best_pos, float* best_vals, float time_per_iter) {

	float* agent_pos = NULL;
	float* agent_val = NULL;

	hipMalloc(&agent_pos, NUM_OF_INDICES * sizeof(float));
	hipMalloc(&agent_val, NUM_OF_AGENTS * sizeof(float));

}

__host__ void GWO(const float* init_pop, const float* init_vals, const int* a, const int* b, 
	float* best_pos, float* best_vals, float time_per_iter) {

	float* agent_pos = NULL;
	float* agent_val = NULL;

	hipMalloc(&agent_pos, NUM_OF_INDICES * sizeof(float));
	hipMalloc(&agent_val, NUM_OF_AGENTS * sizeof(float));

}

__host__ void iGWO(const float* init_pop, const float* init_vals, const int* a, const int* b, 
	float* best_pos, float* best_vals, float time_per_iter) {

	float* agent_pos = NULL;
	float* agent_val = NULL;

	hipMalloc(&agent_pos, NUM_OF_INDICES * sizeof(float));
	hipMalloc(&agent_val, NUM_OF_AGENTS * sizeof(float));

}