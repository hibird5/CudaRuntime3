#include "hip/hip_runtime.h"

#include"alg.cuh"


 __host__ void Diff_ev(const float w, const float p, const float* init_pop,const float* init_vals,const int* a,const  int* b,
	float* best_pos, float* best_vals, float time_per_iter) {
	
	//Initialization
	float* agent_pos = NULL;			
	float* agent_val = NULL;
	float* y_DE_pos = NULL;
	float* y_DE_val = NULL;

	unsigned int* best_index = NULL;
	unsigned int* r = NULL;
	unsigned int* X = NULL;
	unsigned int* HOST_best_index = (unsigned int*)malloc(NUM_OF_AGENTS_HALF * sizeof(unsigned int)); 
	float* Rj;

	hipError_t err = (hipError_t)0;
	hiprandGenerator_t r_gen;
	unsigned int num_of_Ri = 4 * NUM_OF_INDICES;

	hiprandCreateGenerator(&r_gen, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
	hiprandSetPseudoRandomGeneratorSeed(r_gen, 5);

	hipMalloc(&agent_pos, NUM_OF_INDICES * sizeof(float));
	hipMalloc(&agent_val, NUM_OF_AGENTS * sizeof(float));
	hipMalloc(&y_DE_pos, NUM_OF_INDICES * sizeof(float));
	hipMalloc(&y_DE_val, NUM_OF_AGENTS * sizeof(float));
	hipMalloc(&r, num_of_Ri * sizeof(unsigned int));
	hipMalloc(&X, NUM_OF_INDICES * sizeof(unsigned int));
	hipMalloc(&Rj, NUM_OF_INDICES * sizeof(float));
	hipMalloc(&best_index, NUM_OF_AGENTS_HALF* sizeof(unsigned int));

	err = hipGetLastError();

	hipMemcpy(agent_pos, init_pop, NUM_OF_INDICES * sizeof(float), ::hipMemcpyDeviceToDevice);
	hipMemcpy(agent_val, init_vals, NUM_OF_AGENTS * sizeof(float), ::hipMemcpyDeviceToDevice);
	
	error_h(hipGetLastError());

	auto s = std::chrono::high_resolution_clock::now();
	long long iter_time = 0;

	//Loop
	for (int i = 0; i < MAX_ITER; ++i)
	{
		s = std::chrono::high_resolution_clock::now();
		
		//init rands
		hiprandGenerate(r_gen, r, num_of_Ri);
		hiprandGenerate(r_gen, X, NUM_OF_INDICES);
		hiprandGenerateUniform(r_gen, Rj, NUM_OF_INDICES);
		
		//calc new pos
		DE <<<NUM_OF_AGENTS, NUM_OF_DIMS>>> (w, p, a, b, r, X, Rj, best_index, agent_pos, agent_val, y_DE_pos);
		cost_func <<<NUM_OF_AGENTS, DIMS_TO_LOG_HALF >>> (y_DE_pos, y_DE_val);
		compare_two_pop <<<NUM_OF_AGENTS, NUM_OF_DIMS >>> (agent_pos, agent_val, y_DE_pos, y_DE_val);

		//find best pos
		hipDeviceSynchronize();
		searchForBestKernel <<<1, NUM_OF_AGENTS_HALF>>> (agent_val, best_index);
	
		iter_time += std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::high_resolution_clock::now() - s).count();
		
		hipMemcpy(HOST_best_index, best_index, NUM_OF_AGENTS_HALF * sizeof(unsigned int), ::hipMemcpyDeviceToHost);
		hipMemcpy(&best_vals[i], &agent_val[HOST_best_index[0]], sizeof(float), ::hipMemcpyDeviceToDevice);
		
		err = hipGetLastError();
		
		if (err != 0)
			break;
	}

	hipMemcpy(best_pos, &agent_pos[HOST_best_index[0]], NUM_OF_DIMS * sizeof(float), ::hipMemcpyDeviceToDevice);
	time_per_iter = iter_time / MAX_ITER;

	hipFree(agent_pos);
	hipFree(agent_val);
	hipFree(y_DE_pos);
	hipFree(y_DE_val);
	hipFree(r);
	hipFree(X);
	hipFree(Rj);

	error_h(err);
}

__host__ void PSO(const float w, const float c1, const float c2, const float* init_pop, const float* init_vals, const int* a, const int* b,
	float* best_pos, float* best_vals, float time_per_iter) {
	
	//Initialization
	float* agent_pos = NULL;
	float* agent_best_pos = NULL;
	float* agent_val = NULL;
	float* agent_best_val = NULL;

	unsigned int* best_index = NULL;
	float* r;
	unsigned int num_of_ri = 2 * NUM_OF_INDICES;
	unsigned int* HOST_best_index = (unsigned int*)malloc(NUM_OF_AGENTS_HALF * sizeof(unsigned int));

	hipError_t err = (hipError_t)0;
	hiprandGenerator_t r_gen;
	hiprandCreateGenerator(&r_gen, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
	hiprandSetPseudoRandomGeneratorSeed(r_gen, time(NULL));

	hipMalloc(&agent_pos, NUM_OF_INDICES * sizeof(float));
	hipMalloc(&agent_best_pos, NUM_OF_INDICES * sizeof(float));
	hipMalloc(&agent_val, NUM_OF_AGENTS * sizeof(float));
	hipMalloc(&agent_best_val, NUM_OF_AGENTS * sizeof(float));
	hipMalloc(&r, num_of_ri * sizeof(float));
	hipMalloc(&best_index, NUM_OF_AGENTS_HALF * sizeof(unsigned int));


	hipMemcpy(agent_pos, init_pop, NUM_OF_INDICES * sizeof(float), ::hipMemcpyDeviceToDevice);
	hipMemcpy(agent_best_pos, init_pop, NUM_OF_INDICES * sizeof(float), ::hipMemcpyDeviceToDevice);
	hipMemcpy(agent_val, init_vals, NUM_OF_AGENTS * sizeof(float), ::hipMemcpyDeviceToDevice);
	hipMemcpy(agent_best_val, init_vals, NUM_OF_AGENTS * sizeof(float), ::hipMemcpyDeviceToDevice);


	auto s = std::chrono::high_resolution_clock::now();
	long long iter_time = 0;

	//Loop
	for (int i = 0; i < MAX_ITER; ++i)
	{
		s = std::chrono::high_resolution_clock::now();

		//init rand
		hiprandGenerateUniform(r_gen, r, num_of_ri);
		
		//calc new pos
		pso_f <<<NUM_OF_AGENTS, NUM_OF_DIMS >>> (w, c1, c2, a, b, r, best_index, agent_pos, agent_best_pos, agent_val); //V --
		cost_func <<<NUM_OF_AGENTS, DIMS_TO_LOG_HALF >>> (agent_pos, agent_val);
		compare_two_pop <<<NUM_OF_AGENTS, NUM_OF_DIMS >>> (agent_best_pos, agent_best_val, agent_pos, agent_val);

		//find best pos
		hipDeviceSynchronize();
		searchForBestKernel << <1, NUM_OF_AGENTS_HALF >> > (agent_best_val, best_index);
		
		iter_time += std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::high_resolution_clock::now() - s).count();

		hipMemcpy(HOST_best_index, best_index, NUM_OF_AGENTS_HALF * sizeof(unsigned int), ::hipMemcpyDeviceToHost);
		hipMemcpy(&best_vals[i], &agent_best_val[HOST_best_index[0]], sizeof(float), ::hipMemcpyDeviceToHost);

		err = hipGetLastError();

		if (err != 0)
			break;
	}

	hipMemcpy(best_pos, &agent_pos[HOST_best_index[0]], NUM_OF_DIMS * sizeof(float), ::hipMemcpyDeviceToDevice);
	time_per_iter = iter_time / MAX_ITER;

	hipFree(agent_pos);
	hipFree(agent_best_pos);
	hipFree(agent_val);
	hipFree(agent_best_val);
	hipFree(r);
	hipFree(best_index);

	error_h(err);
}

__host__ void FF(const float alfa, const float beta, const float gamma, const float* init_pop, const float* init_vals, const int* a, const int* b,
	float* best_pos, float* best_vals, float time_per_iter) {

	//Initialization
	float* agent_pos = NULL;
	float* agent_val = NULL;
	float* new_pos = NULL;
	float* new_vals = NULL;
	float* cost_func_tmp = NULL;

	unsigned int* best_index = NULL;
	unsigned int* HOST_best_index = (unsigned int*)malloc(NUM_OF_AGENTS_HALF * sizeof(unsigned int));
	float* r;
	unsigned int num_of_uR = POW_OF_AGENTS * NUM_OF_DIMS;

	dim3 agents(NUM_OF_AGENTS, NUM_OF_AGENTS, 1);
	hipError_t err = (hipError_t)0;
	hiprandGenerator_t r_gen;
	hiprandCreateGenerator(&r_gen, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
	hiprandSetPseudoRandomGeneratorSeed(r_gen, time(NULL));
	
	hipMalloc(&agent_pos, NUM_OF_INDICES * sizeof(float));
	hipMalloc(&agent_val, NUM_OF_AGENTS * sizeof(float));
	hipMalloc(&best_index, NUM_OF_AGENTS_HALF * sizeof(unsigned int));
	hipMalloc(&r, num_of_uR * sizeof(float));
	hipMalloc(&new_pos, NUM_OF_AGENTS * NUM_OF_INDICES * sizeof(float));
	hipMalloc(&new_vals, NUM_OF_AGENTS * NUM_OF_AGENTS * sizeof(float));
	hipMalloc(&cost_func_tmp, POW_OF_AGENTS * DIMS_TO_LOG_HALF * sizeof(float));

	hipMemcpy(agent_pos, init_pop, NUM_OF_INDICES * sizeof(float), ::hipMemcpyDeviceToDevice);
	hipMemcpy(agent_val, init_vals, NUM_OF_AGENTS * sizeof(float), ::hipMemcpyDeviceToDevice);

	auto s = std::chrono::high_resolution_clock::now();
	long long iter_time = 0;

	for (int i = 0; i < MAX_ITER; ++i)
	{
		s = std::chrono::high_resolution_clock::now();
		//init rand
		hiprandGenerateNormal(r_gen, r, num_of_uR, 0.0, 0.5);

		//calc new pos
		ffa << <agents, NUM_OF_DIMS >> > (alfa, beta, gamma, a, b, r, agent_pos, new_pos, agent_val);
		cost_func << <agents, DIMS_TO_LOG_HALF >> > (new_pos, new_vals, cost_func_tmp);
		compare_ff_pop << <NUM_OF_AGENTS, NUM_OF_DIMS >> > (agent_pos, agent_val, new_pos, new_vals);

		//find best sol
		hipDeviceSynchronize();
		searchForBestKernel << <1, NUM_OF_AGENTS_HALF >> > (agent_val, best_index);
		
		iter_time += std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::high_resolution_clock::now() - s).count();

		hipMemcpy(HOST_best_index, best_index, NUM_OF_AGENTS_HALF * sizeof(unsigned int), ::hipMemcpyDeviceToHost);
		hipMemcpy(&best_vals[i], &agent_val[HOST_best_index[0]], sizeof(float), ::hipMemcpyDeviceToHost);

		err = hipGetLastError();

		if (err != 0)
			break;

	}

	hipMemcpy(best_pos, &agent_pos[HOST_best_index[0]], NUM_OF_DIMS * sizeof(float), ::hipMemcpyDeviceToDevice);
	time_per_iter = iter_time / MAX_ITER;

	hipFree(agent_pos);
	hipFree(agent_val);
	hipFree(best_index);
	hipFree(r);
	hipFree(new_pos);
	hipFree(new_vals);
	hipFree(cost_func_tmp);

	error_h(err);
}

__host__ void ABC(const float* init_pop,const float* init_vals, const int* a, const int* b, 
	float* best_pos, float* best_vals, float time_per_iter) {

	float* agent_pos = NULL;
	float* agent_val = NULL;

	hipMalloc(&agent_pos, NUM_OF_INDICES * sizeof(float));
	hipMalloc(&agent_val, NUM_OF_AGENTS * sizeof(float));

}

__host__ void GWO(const float* init_pop, const float* init_vals, const int* a, const int* b, 
	float* best_pos, float* best_vals, float time_per_iter) {

	float* agent_pos = NULL;
	float* agent_val = NULL;

	hipMalloc(&agent_pos, NUM_OF_INDICES * sizeof(float));
	hipMalloc(&agent_val, NUM_OF_AGENTS * sizeof(float));



	//float* r_a; float* r_d; float* r;
	//unsigned int* r_w; unsigned int* r_nh;
	//hiprandGenerator_t r_in;
	//unsigned int num_of_uR = pow(NUM_OF_AGENTS, 2);
	//hipMalloc(&r_a, 3 * NUM_OF_AGENTS * sizeof(float));
	//hipMalloc(&r_d, 3 * NUM_OF_AGENTS * sizeof(float));
	//hipMalloc(&r, NUM_OF_INDICES * sizeof(float));

	//hipMalloc(&r_w, NUM_OF_AGENTS * sizeof(unsigned int));
	//hipMalloc(&r_nh, NUM_OF_AGENTS * sizeof(unsigned int));

	//hiprandCreateGenerator(&r_in, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
	//hiprandSetPseudoRandomGeneratorSeed(r_in, time(NULL));

	//hiprandGenerateUniform(r_in, r_a, 6 * NUM_OF_AGENTS);
	//hiprandGenerateUniform(r_in, r, NUM_OF_INDICES);
	//hiprandGenerate(r_in, r_w, 2 * NUM_OF_AGENTS);
	//searchForBestThree << <1, NUM_OF_AGENTS_HALF >> > (agent_val, indice);

	//hipError_t eer;
	//float A = 0;
	//double aaa;
	//auto s = std::chrono::high_resolution_clock::now();;
	//long long e = 0; long long ee = 0; long long eee = 0; long long eeee = 0; long long eeeee = 0; long long eew = 0; long long eeew = 0;
	//for (int i = 0; i < MAX_ITER; ++i)
	//{
	//	A = 2 - 2 * i / MAX_ITER;


	//	//hipMemcpy(indi, indice, NUM_OF_AGENTS * sizeof(unsigned int), ::hipMemcpyDeviceToHost);

	//	//err = hipMemcpy(&best[i], &agent_val[indi[0]], sizeof(float), ::hipMemcpyDeviceToHost);

	//	GWO << < NUM_OF_AGENTS, NUM_OF_DIMS >> > (indice, r_a, a, b, A, agent_pos, X_gwo);

	//	//calc_distances<<<agents,1>>>(agent_pos, dist_gwo);

	//	//iGWO_nh << <agents, NUM_OF_DIMS>> > (r_w, r, a, b, dist_gwo, agent_pos, X_gwo, nh_pos, ind_to_choose);

	//	//	cost_func << <NUM_OF_AGENTS, DIMS_TO_LOG_HALF >> > (nh_pos, nh_val);

	//	cost_func << <NUM_OF_AGENTS, DIMS_TO_LOG_HALF >> > (X_gwo, y_DE_val);

	//	//eer = hipGetLastError();

	//	//hipDeviceSynchronize();
	//	compare_two_pop << <NUM_OF_AGENTS, NUM_OF_DIMS >> > (agent_pos, agent_val, X_gwo, y_DE_val);
	//	searchForBestThree << <1, NUM_OF_AGENTS_HALF >> > (agent_val, indice);


	//	hiprandGenerateUniform(r_in, r_a, 6 * NUM_OF_AGENTS);
	//	hiprandGenerateUniform(r_in, r, NUM_OF_INDICES);
	//	hiprandGenerate(r_in, r_w, 2 * NUM_OF_AGENTS);

	//	err2 = hipMemcpy(&ind[0], &indice[0], 3 * sizeof(unsigned int), ::hipMemcpyDeviceToHost);
	//	err1 = hipMemcpy(&best[i], &agent_val[ind[0]], sizeof(float), ::hipMemcpyDeviceToHost);
	//	err1 = hipMemcpy(pop_vals, agent_val, NUM_OF_AGENTS * sizeof(float), ::hipMemcpyDeviceToHost);

	//	//hipMemcpy(indi, dist_gwo, NUM_OF_AGENTS * sizeof(unsigned int), ::hipMemcpyDeviceToHost);
	//	//for (int i = 0; i <  3; ++i)
	//	//{
	//	//	cout << i << ind[i] << ", " << pop_vals[ind[i]] << endl;
	//	//}
	//	//cout << '\n' << endl;

	//	error_h(hipGetLastError());
	//	//hipMemcpy(indi, dist_gwo, NUM_OF_AGENTS * sizeof(unsigned int), ::hipMemcpyDeviceToHost);
	//	//	for (int i = 0; i <NUM_OF_AGENTS; ++i)
	//	//	{
	//	//		cout << indi[i] << ", " << endl;
	//	//	}
	//	//cout << '\n' << endl;
	//}

}

__host__ void iGWO(const float* init_pop, const float* init_vals, const int* a, const int* b, 
	float* best_pos, float* best_vals, float time_per_iter) {

	float* agent_pos = NULL;
	float* agent_val = NULL;

	hipMalloc(&agent_pos, NUM_OF_INDICES * sizeof(float));
	hipMalloc(&agent_val, NUM_OF_AGENTS * sizeof(float));

	////	dim3 agents(NUM_OF_AGENTS, NUM_OF_AGENTS, 1);

	//float* r_a; float* r_d; float* r;
	//unsigned int* r_w; unsigned int* r_nh;
	//hiprandGenerator_t r_in;
	//unsigned int num_of_uR = pow(NUM_OF_AGENTS, 2);
	//hipMalloc(&r_a, 3 * NUM_OF_AGENTS * sizeof(float));
	//hipMalloc(&r_d, 3 * NUM_OF_AGENTS * sizeof(float));
	//hipMalloc(&r, NUM_OF_INDICES * sizeof(float));

	//hipMalloc(&r_w, NUM_OF_AGENTS * sizeof(unsigned int));
	//hipMalloc(&r_nh, NUM_OF_AGENTS * sizeof(unsigned int));

	//hiprandCreateGenerator(&r_in, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
	//hiprandSetPseudoRandomGeneratorSeed(r_in, time(NULL));

	//hiprandGenerateUniform(r_in, r_a, 6 * NUM_OF_AGENTS);
	//hiprandGenerateUniform(r_in, r, NUM_OF_INDICES);
	//hiprandGenerate(r_in, r_w, 2 * NUM_OF_AGENTS);
	//searchForBestThree << <1, NUM_OF_AGENTS_HALF >> > (agent_val, indice);

	//hipError_t eer;
	//float A = 0;
	//double aaa;
	//auto s = std::chrono::high_resolution_clock::now();;
	//long long e = 0; long long ee = 0; long long eee = 0; long long eeee = 0; long long eeeee = 0; long long eew = 0; long long eeew = 0;
	//for (int i = 0; i < MAX_ITER; ++i)
	//{
	//	A = 2 - 2 * i / MAX_ITER;


	//	//hipMemcpy(indi, indice, NUM_OF_AGENTS * sizeof(unsigned int), ::hipMemcpyDeviceToHost);

	//	//err = hipMemcpy(&best[i], &agent_val[indi[0]], sizeof(float), ::hipMemcpyDeviceToHost);

	//	GWO << < NUM_OF_AGENTS, NUM_OF_DIMS >> > (indice, r_a, a, b, A, agent_pos, X_gwo);

	//	calc_distances << <agents, 1 >> > (agent_pos, tmp_distance, dist_gwo);

	//	iGWO_nh << <agents, NUM_OF_DIMS >> > (r_w, r, a, b, dist_gwo, agent_pos, X_gwo, nh_pos, ind_to_choose);

	//	cost_func << <NUM_OF_AGENTS, DIMS_TO_LOG_HALF >> > (nh_pos, nh_val);

	//	cost_func << <NUM_OF_AGENTS, DIMS_TO_LOG_HALF >> > (X_gwo, y_DE_val);

	//	//eer = hipGetLastError();

	//	//hipDeviceSynchronize();
	//	compare_two_pop << <NUM_OF_AGENTS, NUM_OF_DIMS >> > (agent_pos, agent_val, X_gwo, y_DE_val, nh_pos, nh_val);
	//	searchForBestThree << <1, NUM_OF_AGENTS_HALF >> > (agent_val, indice);


	//	hiprandGenerateUniform(r_in, r_a, 6 * NUM_OF_AGENTS);
	//	hiprandGenerateUniform(r_in, r, NUM_OF_INDICES);
	//	hiprandGenerate(r_in, r_w, 2 * NUM_OF_AGENTS);

	//	err2 = hipMemcpy(&ind[0], &indice[0], 3 * sizeof(unsigned int), ::hipMemcpyDeviceToHost);
	//	err1 = hipMemcpy(&best[i], &agent_val[ind[0]], sizeof(float), ::hipMemcpyDeviceToHost);
	//	err1 = hipMemcpy(pop_vals, agent_val, NUM_OF_AGENTS * sizeof(float), ::hipMemcpyDeviceToHost);

	//	//hipMemcpy(indi, dist_gwo, NUM_OF_AGENTS * sizeof(unsigned int), ::hipMemcpyDeviceToHost);
	//	//for (int i = 0; i <  3; ++i)
	//	//{
	//	//	cout << i << ind[i] << ", " << pop_vals[ind[i]] << endl;
	//	//}
	//	//cout << '\n' << endl;

	//	error_h(hipGetLastError());
	//	//hipMemcpy(indi, dist_gwo, NUM_OF_AGENTS * sizeof(unsigned int), ::hipMemcpyDeviceToHost);
	//	//	for (int i = 0; i <NUM_OF_AGENTS; ++i)
	//	//	{
	//	//		cout << indi[i] << ", " << endl;
	//	//	}
	//	//cout << '\n' << endl;
	//}


}