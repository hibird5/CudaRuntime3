﻿#ifndef __HIPCC__ 
#define __HIPCC__
#endif

#include "h_fce.cuh"
#include "alg.cuh"

#include <stdio.h>
#include <fstream>
#include <iostream>
#include <time.h>

using namespace std;
using namespace hipcub;



int main()
{
	// Device vars
	float* DEV_init_pop;
	float* DEV_init_vals;

	int* DEV_a;
	int* DEV_b;

	float* DEV_best_pos;
	float* DEV_best_vals;

	hipMalloc(&DEV_init_pop, NUM_OF_INDICES * sizeof(float));
	hipMalloc(&DEV_init_vals, NUM_OF_AGENTS * sizeof(float));

	hipMalloc(&DEV_a, NUM_OF_DIMS * sizeof(int));
	hipMalloc(&DEV_b, NUM_OF_DIMS * sizeof(int));

	hipMalloc(&DEV_best_pos, NUM_OF_DIMS * sizeof(float));
	hipMalloc(&DEV_best_vals, MAX_ITER * sizeof(float));
	// Host vars
	float time_iter = .0;
	float* H_best_vals = (float*)malloc(MAX_ITER * sizeof(float));

	hipError_t err = (hipError_t)0;

	
	get_constr <<<NUM_OF_DIMS, 1 >>> (lo, hi, DEV_a, DEV_b);
	init_pop_pos <<<NUM_OF_AGENTS, NUM_OF_DIMS >>> (DEV_init_pop, DEV_a, DEV_b, (unsigned long)time(NULL));
	cost_func <<<NUM_OF_AGENTS, DIMS_TO_LOG_HALF >>> (DEV_init_pop, DEV_init_vals);

	err = hipGetLastError();
	error_h(err);

//	Diff_ev(0.4, 0.7, DEV_init_pop, DEV_init_vals, DEV_a, DEV_b, DEV_best_pos, DEV_best_vals, time_iter);
//	PSO(0.1, 0.25, 2, DEV_init_pop, DEV_init_vals, DEV_a, DEV_b, DEV_best_pos, DEV_best_vals, time_iter);
//	FF(1, 1, 0.1, DEV_init_pop, DEV_init_vals, DEV_a, DEV_b, DEV_best_pos, DEV_best_vals, time_iter);
//	GWO(DEV_init_pop, DEV_init_vals, DEV_a, DEV_b, DEV_best_pos, DEV_best_vals, time_iter);
	iGWO(DEV_init_pop, DEV_init_vals, DEV_a, DEV_b, DEV_best_pos, DEV_best_vals, time_iter);



	hipMemcpy(H_best_vals, DEV_best_vals, MAX_ITER * sizeof(unsigned int), ::hipMemcpyDeviceToHost);

	for (int i = 0; i < MAX_ITER; ++i)
	{
		cout << i << ".   " << H_best_vals[i] << ", " << endl;
		//for (int j = 0; j < NUM_OF_DIMS; ++j)
		//{
		//	cout << pop_back[i * NUM_OF_DIMS + j] << ", ";
		//}
		//	cout << endl;
	}
	cout << '\n' << endl;
	
	hipFree(DEV_init_pop);
	hipFree(DEV_init_vals);
	hipFree(DEV_a);
	hipFree(DEV_b);
	hipFree(DEV_best_pos);
	hipFree(DEV_best_vals);

	return 0;

}