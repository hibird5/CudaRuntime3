#include "hip/hip_runtime.h"
#ifndef __HIPCC__ 
#define __HIPCC__
#endif


#include "h_fce.cuh"

#include <stdio.h>
#include <time.h>

using namespace std;
using namespace hipcub;






__global__ void searchForBestKernel(volatile float* objectiveValues, size_t* indices)
{
	unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
	__shared__ float oVs[num_of_agents];
	__shared__ unsigned int ind[num_of_agents];

	oVs[id] = objectiveValues[id];
	oVs[id + num_of_agents_half] = objectiveValues[id + num_of_agents_half];
	ind[id] = id;
	ind[id + num_of_agents_half] = id + num_of_agents_half;
	__syncthreads();
	unsigned int step = num_of_agents_half;

#pragma unroll
	for (int i = 0; i < num_of_runs; ++i)
	{
		ind[id] = ((oVs[ind[id + step]] < oVs[ind[id]]) ? ind[id + step] : ind[id]);
		step >>= 1;
		__syncthreads();
	}
	indices[id] = ind[id];
	__syncthreads();
}



 

int main()
{
	//device init
	float* agent_pos = NULL;
	float* agent_val = NULL;
	size_t* indice = NULL;
	int* a = NULL;
	int* b = NULL;
	hiprandGenerator_t pseudo_rand;
	float* best_sol_a = NULL;
	float* y_DE;
	float* y_DE_val;
	float* best_DE;
	float* ff_new_poss;
	float* ff_new_vals;
	size_t* best_de;

	float* agent_best_pso;
	float* agent_best_pso_v;
	size_t* best_pso;

	hipMalloc(&agent_pos, num_of_indices * sizeof(float));
	hipMalloc(&agent_val, num_of_agents * sizeof(float));

	hipMalloc(&y_DE, num_of_indices * sizeof(float));
	hipMalloc(&y_DE_val, num_of_agents * sizeof(float));
	hipMalloc(&best_DE, max_iter * sizeof(float));		//vals
	hipMalloc(&best_de, sizeof(size_t));				//indice

	hipMalloc(&agent_best_pso, num_of_indices * sizeof(float));
	hipMalloc(&agent_best_pso_v, num_of_agents * sizeof(float));
	hipMalloc(&best_pso, max_iter * sizeof(float));		//vals

	hipMalloc(&ff_new_poss, num_of_agents*num_of_indices * sizeof(float));
	hipMalloc(&ff_new_vals, num_of_agents * num_of_agents * sizeof(float));

	hipMalloc(&indice, num_of_agents * sizeof(size_t));
	hipMalloc(&best_sol_a, num_of_best_indices * sizeof(float));
	hipMalloc(&a, num_of_dims * sizeof(int));
	hipMalloc(&b, num_of_dims * sizeof(int));

	//host init
	float* pop_back = NULL;
	float* pop_vals = NULL;
	float* best = NULL;
	size_t* ind;

	pop_back = (float*)malloc(num_of_indices * sizeof(float));
	pop_vals = (float*)malloc(num_of_agents * sizeof(float));
	best = (float*)malloc(max_iter * sizeof(float));
	ind = (size_t*)malloc(num_of_agents * sizeof(size_t));

	// prog
	get_constr << <num_of_dims, 1 >> > (-100, 100, a, b);
	init_pop_pos << <num_of_agents, num_of_dims >> > (agent_pos, a, b, time(NULL));
	cost_func << <num_of_agents, 1 >> > (agent_pos, agent_val);

	hipMemcpy(pop_vals, agent_val, num_of_agents * sizeof(float), ::hipMemcpyDeviceToHost);

	searchForBestKernel << <best_bl_th, best_bl_th >> > (agent_val, indice);
	hipMemcpy(ind, indice, num_of_agents * sizeof(size_t), ::hipMemcpyDeviceToHost);


	hipError_t err; hipError_t err1; hipError_t err2;
	//err = hipMemcpy(best_de, &indice[0], sizeof(size_t), ::hipMemcpyDeviceToDevice);

	//DE start

	//	time_t start = clock();
	//for (int i = 0; i < max_iter; ++i)
	//{
	//	DE << <num_of_agents, num_of_dims >> > (0.4, 0.7, a, b, time(NULL), indice, agent_pos, agent_val, y_DE);
	//	cost_func << <num_of_agents, 1 >> > (y_DE, y_DE_val);
	//	compare_two_pop << <num_of_agents, 1 >> > (agent_pos, agent_val, y_DE, y_DE_val);
	//	searchForBestKernel << <best_bl_th, best_bl_th>> > (agent_val, indice);
	//	hipMemcpy(ind, indice, num_of_agents * sizeof(size_t), ::hipMemcpyDeviceToHost);
	//	err = hipMemcpy(&best[i], &agent_val[ind[0]], sizeof(float), ::hipMemcpyDeviceToHost);
	//	err1 = hipMemcpy(best_de, &indice[0], sizeof(size_t), ::hipMemcpyDeviceToDevice);
	//}

	//DE end
	
	//PSO start
	
	// 
	//agent_best_pso = agent_pos;
	//agent_best_pso_v = agent_val;
	//for (int i = 0; i < max_iter; ++i)
	//{
	//	pso_f << <num_of_agents, num_of_dims >> > (0.1, 0.25, 2, a, b, time(NULL), indice, agent_pos, agent_best_pso, agent_val);
	//	cost_func << <num_of_agents, 1 >> > (agent_pos, agent_val);
	//	compare_two_pop << <num_of_agents, 1 >> > (agent_best_pso, agent_best_pso_v, agent_pos, agent_val);
	//	searchForBestKernel << <best_bl_th, best_bl_th>> > (agent_best_pso_v, indice);
	//	hipMemcpy(ind, indice, num_of_agents * sizeof(size_t), ::hipMemcpyDeviceToHost);
	//	err = hipMemcpy(&best[i], &agent_best_pso_v[ind[0]], sizeof(float), ::hipMemcpyDeviceToHost);
	//}
	
	//PSO end
	// 
	//FF start

	for (int i = 0; i < max_iter; ++i)
	{
		ffa << <num_of_agents, num_of_agents >> > (1, 1, 0.01, a, b, time(NULL), agent_pos, ff_new_poss, agent_val);
		//	ke� po ffa zavolam MemCpy ???
		err2 = hipMemcpy(pop_vals, agent_val, num_of_agents * sizeof(float), ::hipMemcpyDeviceToHost);
		
		cost_func << <pow_of_agents, 1 >> > (ff_new_poss, ff_new_vals);
		
		compare_ff_pos << <num_of_agents, 1 >> > (agent_pos, agent_val, ff_new_poss, ff_new_vals);
		
		searchForBestKernel << <best_bl_th, best_bl_th>> > (agent_val, indice);

		//err1 = hipMemcpy(ind, indice, num_of_agents * sizeof(size_t), ::hipMemcpyDeviceToHost);
		/*err = hipMemcpy(&best[i], &agent_val[ind[0]], sizeof(float), ::hipMemcpyDeviceToHost);
		err1 = hipMemcpy(best_de, &indice[0], sizeof(size_t), ::hipMemcpyDeviceToDevice);*/
	}

//FF end

	
		err2 = hipMemcpy(pop_vals, agent_pos, num_of_agents * sizeof(float), ::hipMemcpyDeviceToHost);
		for (int i = 0; i < num_of_agents; ++i)
		{
			cout << pop_vals[i] << ", " << endl;
		}

	//err = hipMemcpy(pop_back, agent_pos, num_of_indices * sizeof(float), ::hipMemcpyDeviceToHost);
	//for (int i = 0; i < num_of_agents; ++i)
	//{
	//	cout << i << " ";
	//	for (int j = 0; j < num_of_dims; ++j)
	//	{
	//		cout << pop_back[i * num_of_dims + j] << ", ";
	//	}
	//	cout << '\n' << endl;
	//}
	//cout << '\n' << endl;

	cout << ind[0] << ", " << endl;

	for (int i = 0; i < max_iter; ++i)
	{
		cout << best[i] << ", " << endl;
	}
	cout << '\n' << endl;


	free(pop_back);
	free(pop_vals);
	free(best);

	hipFree(agent_pos);
	hipFree(agent_val);
	hipFree(indice);
	hipFree(a);
	hipFree(b);
	hipFree(best_sol_a);
	hipFree(y_DE);
	hipFree(y_DE_val);
	hipFree(best_DE);
	hipFree(best_de);
	

	return 0;
}


//hiprand array
//float* r = NULL;
//hipMalloc(&r, num_of_indices * sizeof(float));
//hiprandCreateGenerator(&pseudo_rand, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
//hiprandSetPseudoRandomGeneratorSeed(pseudo_rand, 1);
//hiprandGenerateUniform(pseudo_rand, r, num_of_indices);
//hiprandGenerateUniform(pseudo_rand, r, num_of_indices);
//hiprandGenerateUniform(pseudo_rand, r, num_of_indices);

