#include "hip/hip_runtime.h"
#ifndef __HIPCC__ 
#define __HIPCC__
#endif


#include "h_fce.cuh"

#include <stdio.h>
#include <time.h>

using namespace std;
using namespace hipcub;



__global__ void searchForBestKernel(volatile float* objectiveValues, unsigned int* indices)
{
	unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
	__shared__ float oVs[num_of_agents];
	__shared__ unsigned int ind[num_of_agents];

	oVs[id] = objectiveValues[id];
	oVs[id + num_of_agents_half] = objectiveValues[id + num_of_agents_half];
	ind[id] = id;
	ind[id + num_of_agents_half] = id + num_of_agents_half;
	__syncthreads();
	unsigned int step = num_of_agents_half;

#pragma unroll
	for (int i = 0; i < num_of_runs; ++i)
	{
		ind[id] = ((oVs[ind[id + step]] < oVs[ind[id]]) ? ind[id + step] : ind[id]);
		step >>= 1;
		__syncthreads();
	}
	indices[id] = ind[id];
	__syncthreads();
}

__global__ void searchForBestThree(volatile float* objectiveValues, unsigned int* best_three)
{
	best_three[0] = num_of_agents;
	best_three[1] = num_of_agents;

	unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
	__shared__ float oVs[num_of_agents];
	__shared__ unsigned int ind[num_of_agents];

	for (int j = 0; j < 3; j++) {

		oVs[id] = objectiveValues[id];
		oVs[id + num_of_agents_half] = objectiveValues[id + num_of_agents_half];
		ind[id] = id;
		ind[id + num_of_agents_half] = id + num_of_agents_half;
		__syncthreads();
		unsigned int step = num_of_agents_half;

#pragma unroll
		for (int i = 0; i < num_of_runs; ++i)
		{	
			unsigned int ind_s = id + step;
			if (ind[id] == best_three[0] || ind[id] == best_three[1])
				ind[id] = ind[ind_s];
			else if(ind[ind_s] == best_three[0] || ind[ind_s] == best_three[1])
				ind[id] = ind[id];
			else
				ind[id] = ((oVs[ind[ind_s]] < oVs[ind[id]]) ? ind[ind_s] : ind[id]);
			step >>= 1;
			__syncthreads();
		}
		best_three[0] = (j == 0) ? ind[id] : best_three[0];
		best_three[1] = (j == 1) ? ind[id] : best_three[1];
		best_three[2] = (j == 2) ? ind[id] : best_three[2];
		__syncthreads();
	}
}

 

int main()
{
	//device init
	float* agent_pos = NULL;
	float* agent_val = NULL;
	unsigned int* indice = NULL;
	int* a = NULL;
	int* b = NULL;
	float* best_sol_a = NULL;
	float* y_DE;
	float* y_DE_val;
	float* best_DE;
	float* ff_new_poss;
	float* ff_new_vals;
	unsigned int* best_de;

	float* agent_best_pso;
	float* agent_best_pso_v;
	unsigned int* best_pso;
	float* X_gwo;
	float* nh_pos;
	float* nh_val;
	float* dist_gwo;
	unsigned int* ind_to_choose;
	float* cost_func_tmp;

	hipMalloc(&X_gwo, num_of_indices * sizeof(float));
	hipMalloc(&nh_pos, num_of_indices * sizeof(float));
	hipMalloc(&nh_val, num_of_agents * sizeof(float));
	hipMalloc(&dist_gwo, pow_of_agents * sizeof(float));
	hipMalloc(&ind_to_choose, pow_of_agents * sizeof(unsigned int));
	hipMalloc(&agent_pos, num_of_indices * sizeof(float));
	hipMalloc(&agent_val, num_of_agents * sizeof(float));
	
	hipMalloc(&y_DE, num_of_indices * sizeof(float));
	hipMalloc(&y_DE_val, num_of_agents * sizeof(float));
	hipMalloc(&best_DE, max_iter * sizeof(float));		//vals
	hipMalloc(&best_de, sizeof(unsigned int));				//indice

	hipMalloc(&agent_best_pso, num_of_indices * sizeof(float));
	hipMalloc(&agent_best_pso_v, num_of_agents * sizeof(float));
	hipMalloc(&best_pso, max_iter * sizeof(float));		//vals

	hipMalloc(&ff_new_poss, num_of_agents*num_of_indices * sizeof(float));
	hipMalloc(&ff_new_vals, num_of_agents * num_of_agents * sizeof(float));

	hipMalloc(&indice, num_of_agents * sizeof(unsigned int));
	hipMalloc(&best_sol_a, num_of_best_indices * sizeof(float));
	hipMalloc(&a, num_of_dims * sizeof(int));
	hipMalloc(&b, num_of_dims * sizeof(int));

	hipMalloc(&cost_func_tmp, pow_of_agents * dims_to_log_half * sizeof(float));

	//host init
	float* pop_back = NULL;
	float* pop_vals = NULL;
	float* best = NULL;
	float* ff = NULL;
	unsigned int* ind;
	unsigned int* indi;
	ff = (float*)malloc(num_of_agents * num_of_agents * num_of_dims * sizeof(float));
	pop_back = (float*)malloc(num_of_indices * sizeof(float));
	pop_vals = (float*)malloc(num_of_agents * sizeof(float));
	best = (float*)malloc(max_iter * sizeof(float));
	ind = (unsigned int*)malloc(num_of_agents * sizeof(unsigned int));
	indi = (unsigned int*)malloc(num_of_agents * sizeof(unsigned int));
	// prog

	get_constr << <num_of_dims, 1 >> > (lo, hi, a, b);
	init_pop_pos << <num_of_agents, num_of_dims >> > (agent_pos, a, b, (unsigned long)time(NULL));
	cost_func << <num_of_agents, dims_to_log_half >> > (agent_pos, agent_val, cost_func_tmp);
	//thrust::sort(thrust::device, agent_val, agent_val);

	//
	//hipMemcpy(pop_vals, agent_val, num_of_agents * sizeof(float), ::hipMemcpyDeviceToHost);
	//	for (int i = 0; i < num_of_agents; ++i)
	//	{
	//		cout << pop_vals[i] << ", " << endl;
	//	}
	 //hipMemcpy(pop_vals, agent_val, num_of_agents * sizeof(float), ::hipMemcpyDeviceToHost);
	 //for (int i = 0; i < num_of_agents; ++i)
	 //{
		// cout << i << ".    " << pop_vals[i] << ", " << endl;
	 //}
	 //cout << endl;

	 //hipMemcpy(pop_back, agent_pos, num_of_indices * sizeof(float), ::hipMemcpyDeviceToHost);
	 //for (int i = 0; i < num_of_agents; ++i)
	 //{
 	//	cout << i << " ";
 	//	for (int j = 0; j < num_of_dims; ++j)
 	//	{
 	//		cout << pop_back[i * num_of_dims + j] << ", ";
 	//	}
 	//	cout << '\n' << endl;
	 //}
	 //cout << '\n' << endl;
	//searchForBestKernel << <1, num_of_agents_half >> > (agent_val, indice);

	hipError_t err; hipError_t err1; hipError_t err2;
	//err = hipMemcpy(best_de, &indice[0], sizeof(unsigned int), ::hipMemcpyDeviceToDevice);

	//thrust::sort(thrust::device, agent_val, agent_val + num_of_agents);
	err = hipGetLastError();

	//DE start
	
	//unsigned int* r;
	//unsigned int* X;
	//float* Rj;
	//hiprandGenerator_t r_int;
	//unsigned int num_of_Ri = 4 * num_of_indices;
	//hipMalloc(&r, num_of_Ri * sizeof(unsigned int));
	//hipMalloc(&X, num_of_indices * sizeof(unsigned int));
	//hipMalloc(&Rj, num_of_indices * sizeof(float));
	//
	//hiprandCreateGenerator(&r_int, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
	//hiprandSetPseudoRandomGeneratorSeed(r_int, time(NULL));
	//
	//for (int i = 0; i < max_iter; ++i)
	//{
	//	hiprandGenerate(r_int, r, num_of_Ri);
	//	hiprandGenerate(r_int, X, num_of_indices);
	//	hiprandGenerateUniform(r_int, Rj, num_of_indices);
	//
	//	DE << <  num_of_agents, num_of_dims>> > (0.4, 0.7, a, b, r, X, Rj,
	//											indice, agent_pos, agent_val, y_DE);
	//	hipDeviceSynchronize();
	//	cost_func << <1, num_of_agents >> > (y_DE, y_DE_val);
	//	hipDeviceSynchronize();
	//	compare_two_pop << <num_of_agents, num_of_dims >> > (agent_pos, agent_val, y_DE, y_DE_val);
	//	hipDeviceSynchronize();
	//	searchForBestKernel << <1, num_of_agents_half>> > (agent_val, indice);
	//
	//	hipMemcpy(ind, indice, num_of_agents * sizeof(unsigned int), ::hipMemcpyDeviceToHost);
	//	err = hipMemcpy(&best[i], &agent_val[ind[0]], sizeof(float), ::hipMemcpyDeviceToHost);
	//	err1 = hipMemcpy(best_de, &indice[0], sizeof(unsigned int), ::hipMemcpyDeviceToDevice);
	//}
	
	//DE end
	
	//FF start

	//dim3 agents(num_of_agents, num_of_agents, 1);
	//float* r_a;
	//hiprandGenerator_t r_in;
	//unsigned int num_of_uR = pow_of_agents * num_of_dims;
	//hipMalloc(&r_a, num_of_uR * sizeof(float));
	//
	//hiprandCreateGenerator(&r_in, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
	//hiprandSetPseudoRandomGeneratorSeed(r_in, time(NULL));
	//
	//
	////searchForBestFF<< <num_of_agents_half, num_of_agents>> > (ff_new_vals, indice);
	//for (int i = 0; i < max_iter; ++i)
	//{
	//	hiprandGenerateNormal(r_in, r_a, num_of_uR, 0.0, 0.5);
	//	ffa << <agents, num_of_dims >> > (1, 1, 1/10, a, b, r_a, agent_pos, ff_new_poss, agent_val);
	//	hipDeviceSynchronize();
	//	cost_func << <agents, dims_to_log_half>> > (ff_new_poss, ff_new_vals, cost_func_tmp);
	//
	//
	//	hipDeviceSynchronize();
	//	compare_ff_pos << <num_of_agents, num_of_dims >> > (agent_pos, agent_val, ff_new_poss, ff_new_vals);
	//	hipDeviceSynchronize();
	//	
	//	//hipMemcpy(pop_vals, agent_val, num_of_agents * sizeof(float), ::hipMemcpyDeviceToHost);
	//	//for (int i = 0; i < num_of_agents; ++i)
	//	//{
	//	//	cout << i << ".    " << pop_vals[i] << ", " << endl;
	//	//}
	//	//cout << endl;
	//
	//	hipMemcpy(pop_back, ff_new_poss, num_of_indices * sizeof(float), ::hipMemcpyDeviceToHost);
	//	for (int i = 0; i < num_of_agents; ++i)
	//	{
	//		cout << i << " ";
	//		for (int j = 0; j < num_of_dims; ++j)
	//		{
	//			cout << pop_back[i * num_of_dims + j] << ", ";
	//		}
	//		cout << '\n' << endl;
	//	}
	//	cout << '\n' << endl;
	//
	//	searchForBestKernel << <1, num_of_agents_half >> > (agent_val, indice);
	//
	//	hipMemcpy(ind, indice, num_of_agents * sizeof(unsigned int), ::hipMemcpyDeviceToHost);
	//	hipMemcpy(&best[i], &agent_val[ind[0]], sizeof(float), ::hipMemcpyDeviceToHost);
	//	err = hipGetLastError();
	//}

	//FF end

	//PSO start
	
	//float* r;
	//unsigned int num_of_ri = 2 * num_of_indices;
	//hiprandGenerator_t r_int;
	//hipMalloc(&r, num_of_ri * sizeof(float));
	//
	//hiprandCreateGenerator(&r_int, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
	//hiprandSetPseudoRandomGeneratorSeed(r_int, time(NULL));
	//
	//agent_best_pso = agent_pos;
	//agent_best_pso_v = agent_val;
	//for (int i = 0; i < max_iter; ++i)
	//{
	//	hiprandGenerateUniform(r_int, r, num_of_ri);
	//	pso_f << <  num_of_agents, num_of_dims>> > (0.1, 0.25, 2, a, b, r, indice, agent_pos, agent_best_pso, agent_val);
	//	cost_func << <num_of_agents, dims_to_log_half >> > (agent_pos, agent_val);
	//	compare_two_pop << <num_of_agents, num_of_dims >> > (agent_best_pso, agent_best_pso_v, agent_pos, agent_val);
	//	searchForBestKernel << <1, num_of_agents_half >> > (agent_best_pso_v, indice);
	//	hipMemcpy(ind, indice, num_of_agents * sizeof(unsigned int), ::hipMemcpyDeviceToHost);
	//	err = hipMemcpy(&best[i], &agent_best_pso_v[ind[0]], sizeof(float), ::hipMemcpyDeviceToHost);
	//}
	
	//PSO end



	//iGWO start
/*
	dim3 agents(num_of_agents, num_of_agents, 1);

	float* r_a; float* r_d; float* r;
	unsigned int* r_w; unsigned int* r_nh;
	hiprandGenerator_t r_in;
	unsigned int num_of_uR = pow(num_of_agents, 2);
	hipMalloc(&r_a, 3 * num_of_agents * sizeof(float));
	hipMalloc(&r_d, 3 * num_of_agents * sizeof(float));
	hipMalloc(&r, num_of_indices * sizeof(float));

	hipMalloc(&r_w, num_of_agents* sizeof(unsigned int));
	hipMalloc(&r_nh, num_of_agents * sizeof(unsigned int));

	hiprandCreateGenerator(&r_in, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
	hiprandSetPseudoRandomGeneratorSeed(r_in, time(NULL));

	hiprandGenerateUniform(r_in, r_a, 6 * num_of_agents);
	hiprandGenerateUniform(r_in, r, num_of_indices);
	hiprandGenerate(r_in, r_w, 2 * num_of_agents);

	hipError_t eer;
	float A = 0;
	double aaa;
	auto s = std::chrono::high_resolution_clock::now();;
	long long e = 0; long long ee = 0; long long eee = 0; long long eeee = 0; long long eeeee = 0; long long eew = 0; long long eeew = 0;
	for (int i = 0; i < max_iter; ++i) 
	{
		A = 2 - 2 * i / max_iter;

		searchForBestThree << <1, num_of_agents_half >> > (agent_val, indice);

		//hipMemcpy(indi, indice, num_of_agents * sizeof(unsigned int), ::hipMemcpyDeviceToHost);

		//err = hipMemcpy(&best[i], &agent_val[indi[0]], sizeof(float), ::hipMemcpyDeviceToHost);

		GWO << < num_of_agents, num_of_dims>> > (indice, r_a, a, b, A, agent_pos, X_gwo);

		calc_distances<<<agents,1>>>(agent_pos, dist_gwo);

		iGWO_nh << <agents, num_of_dims>> > (r_w, r, a, b, dist_gwo, agent_pos, X_gwo, nh_pos, ind_to_choose);

		cost_func << <num_of_agents, dims_to_log_half >> > (nh_pos, nh_val);
	
		cost_func << <num_of_agents, dims_to_log_half>> > (X_gwo, agent_val);
		
		eer = hipGetLastError();

		//hipDeviceSynchronize();
		compare_two_pop << <num_of_agents, num_of_dims >> > (agent_pos, agent_val, X_gwo, agent_val, nh_pos, nh_val);
		hiprandGenerateUniform(r_in, r_a, 6 * num_of_agents);
		hiprandGenerateUniform(r_in, r, num_of_indices);
		hiprandGenerate(r_in, r_w, 2 * num_of_agents);


		hipMemcpy(indi, dist_gwo, num_of_agents * sizeof(unsigned int), ::hipMemcpyDeviceToHost);
			for (int i = 0; i <num_of_agents; ++i)
			{
				cout << indi[i] << ", " << endl;
			}
		cout << '\n' << endl;
	}*/
	/*	s = std::chrono::high_resolution_clock::now();
		eeew += std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::high_resolution_clock::now() - s).count();
	e /= 1000; ee /= 1000; eee /= 1000; eeee /= 1000; eeeee /= 1000; eeew /= 1000; eew /= 1000;*/
	//igwo end

	//abc start

	float* r;
	unsigned int* rI;
	unsigned int* abbadon_dec;
	unsigned int* index_to_rns;
	hiprandGenerator_t r_int;
	hipMalloc(&r, num_of_indices * sizeof(float));
	hipMalloc(&rI, num_of_agents * sizeof(float));
	hipMalloc(&index_to_rns, num_of_agents * sizeof(unsigned int));
	hipMalloc(&abbadon_dec, num_of_agents * sizeof(unsigned int));
	hiprandCreateGenerator(&r_int, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
	hiprandSetPseudoRandomGeneratorSeed(r_int, time(NULL));
	

	unsigned int abbadon_val = max_iter / 2;

	for (int i = 0; i < max_iter; ++i)
	{
		hiprandGenerate(r_int, rI, num_of_agents);
		hiprandGenerateUniform(r_int, r, num_of_indices);

		abc_rns << <num_of_agents, num_of_dims >> > (agent_pos, agent_best_pso, a, b, r, rI);
		cost_func << <num_of_agents, dims_to_log_half >> > (agent_best_pso, agent_best_pso_v, cost_func_tmp);
		compare_two_pop << <num_of_agents, num_of_dims >> > (agent_pos, agent_val, agent_best_pso, agent_best_pso_v, abbadon_dec);
		probability_selection << <1, num_of_agents >> > (agent_val, r, index_to_rns);
		err = hipGetLastError();
		//err2 = hipMemcpy(ind, index_to_rns, num_of_agents * sizeof(unsigned int), ::hipMemcpyDeviceToHost);
		//for (int i = 0; i < num_of_agents; ++i)
		//{
		//	cout << ind[i] << ", " << endl;
		//}

		hiprandGenerate(r_int, rI, num_of_agents);
		hiprandGenerateUniform(r_int, r, num_of_indices);
		abc_rns << <num_of_agents, num_of_dims >> > (agent_pos, agent_best_pso, index_to_rns, a, b, r, rI);
		err = hipGetLastError();
		cost_func << <num_of_agents, dims_to_log_half >> > (agent_best_pso, agent_best_pso_v, cost_func_tmp);
		compare_two_pop << <num_of_agents, num_of_dims >> > (agent_pos, agent_val, agent_best_pso, agent_best_pso_v, abbadon_dec);

		hiprandGenerateUniform(r_int, r, num_of_indices);
		err = hipGetLastError();
		scout_phase<<<num_of_agents, num_of_dims>>>(abbadon_dec, 2, a, b, r, agent_pos);
		err2 = hipMemcpy(pop_back, agent_pos, num_of_agents * sizeof(float), ::hipMemcpyDeviceToHost);
		err = hipGetLastError();
		cost_func << <num_of_agents, dims_to_log_half >> > (agent_pos, agent_val, cost_func_tmp);
		err = hipGetLastError();
		searchForBestKernel << <1, num_of_agents_half>> > (agent_val, indice);
		err2 = hipMemcpy(ind, indice, num_of_agents * sizeof(unsigned int), ::hipMemcpyDeviceToHost);
		err1 = hipMemcpy(&best[i], &agent_val[ind[0]], sizeof(float), ::hipMemcpyDeviceToHost);

		err = hipGetLastError();

	}



	cout << ind[0] << ", " << endl;

	for (int i = 0; i < max_iter; ++i)
	{
		cout << best[i] << ", " << endl;
	}
	cout << '\n' << endl;


	free(pop_back);
	free(pop_vals);
	free(best);

	hipFree(agent_pos);
	hipFree(agent_val);
	hipFree(indice);
	hipFree(a);
	hipFree(b);
	hipFree(best_sol_a);
	hipFree(y_DE);
	hipFree(y_DE_val);
	hipFree(best_DE);
	hipFree(best_de);
	

	return 0;
}


//hiprand array
//float* r = NULL;
//hipMalloc(&r, num_of_indices * sizeof(float));
//hiprandCreateGenerator(&pseudo_rand, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
//hiprandSetPseudoRandomGeneratorSeed(pseudo_rand, 1);
//hiprandGenerateUniform(pseudo_rand, r, num_of_indices);
//hiprandGenerateUniform(pseudo_rand, r, num_of_indices);
//hiprandGenerateUniform(pseudo_rand, r, num_of_indices);


	//err = hipMemcpy(pop_back, agent_pos, num_of_indices * sizeof(float), ::hipMemcpyDeviceToHost);
	//for (int i = 0; i < num_of_agents; ++i)
	//{
	//	cout << i << " ";
	//	for (int j = 0; j < num_of_dims; ++j)
	//	{
	//		cout << pop_back[i * num_of_dims + j] << ", ";
	//	}
	//	cout << '\n' << endl;
	//}
	//cout << '\n' << endl;
	
		//err2 = hipMemcpy(pop_vals, agent_pos, num_of_agents * sizeof(float), ::hipMemcpyDeviceToHost);
		//for (int i = 0; i < num_of_agents; ++i)
		//{
		//	cout << pop_vals[i] << ", " << endl;
		//}
		
	//int* gg;
	//hipMalloc(&gg, 4 * sizeof(int));
	//int *gge = (int*)malloc(4* sizeof(int));
	//tet << <agents, block>> > (gg);
	//err =hipMemcpy(gge, gg, 4 * sizeof(int), ::hipMemcpyDeviceToHost);
	//for (int i = 0; i < 4; ++i)
	//{
	//	cout << gge[i] << endl;
	//}